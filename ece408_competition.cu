#include "hip/hip_runtime.h"

/*****************************************************************************
 * Copyright (C) 2013 x265 project
 *
 * Authors: Gopu Govindaswamy <gopu@govindaswamy.org>
 *          Mandar Gurav <mandar@multicorewareinc.com>
 *          Mahesh Pittala <mahesh@multicorewareinc.com>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02111, USA.
 *
 * This program is also available under a commercial proprietary license.
 * For more information, contact us at licensing@multicorewareinc.com.
 *****************************************************************************/

#include "ece408_competition.h"
#include "primitives.h"
#include "test/intrapredharness.h"
#include "cpu.h"
#include "TLibCommon/TComRom.h"
#include "TLibEncoder/TEncCfg.h"

#include "input/input.h"
#include "output/output.h"
#include "common.h"
#include "x265.h"
#include "getopt.h"
#include "PPA/ppa.h"

#include "encoder.h"
#include "TLibCommon/TComYuv.h"
#include "TLibCommon/TComPic.h"
#include "TLibCommon/TComPicYuv.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <fstream>
#include <algorithm>

#include "kernel.cu"

//Define this to verify the student intra prediction against the reference version
#define VERIFY

using namespace x265;

void ece408_setup_pic(x265_encoder *enc, TComPic* &pic_ptr, int cu_size);
void ece408_load_pic(x265_encoder *enc, x265_picture *pic_in, TComPic *pic);
ece408_intra_pred_result *ece408_competition_ref(Encoder *encoder, x265_picture *pics_in, TComPic* &pic, int num_frames);
ece408_intra_pred_result *ece408_competition(ece408_frame *imgs, int num_frames);
bool ece408_compare(ece408_intra_pred_result *ref, ece408_intra_pred_result *student, int num_frames);

Pel *refAbove1, *refAbove2, *refLeft1, *refLeft2;
Pel*      predBuf;
int       predBufStride;
int       predBufHeight;
TComYuv pred_yuv;
TComYuv orig_yuv;
TComSPS sps;
TComPPS pps;
x265_param param;

ALIGN_VAR_32(Pel, tmp[33 * 64 * 64]);
ALIGN_VAR_32(Pel, buf_trans[64 * 64]);

static const char short_options[] = "o:f:F:r:i:b:s:q:m:hwV";
static const struct option long_options[] =
{
#if HIGH_BIT_DEPTH
    { "depth",          required_argument, NULL, 0 },
#endif
    { "help",                 no_argument, NULL, 'h' },
    { "cpuid",          required_argument, NULL, 0 },
    { "threads",        required_argument, NULL, 0 },
    { "preset",         required_argument, NULL, 'p' },
    { "tune",           required_argument, NULL, 't' },
    { "frame-threads",  required_argument, NULL, 'F' },
    { "log",            required_argument, NULL, 0 },
    { "csv",            required_argument, NULL, 0 },
    { "y4m",                  no_argument, NULL, 0 },
    { "no-progress",          no_argument, NULL, 0 },
    { "output",         required_argument, NULL, 'o' },
    { "input",          required_argument, NULL, 0 },
    { "input-depth",    required_argument, NULL, 0 },
    { "input-res",      required_argument, NULL, 0 },
    { "input-csp",      required_argument, NULL, 0 },
    { "fps",            required_argument, NULL, 0 },
    { "frame-skip",     required_argument, NULL, 0 },
    { "frames",         required_argument, NULL, 'f' },
    { "recon",          required_argument, NULL, 'r' },
    { "recon-depth",    required_argument, NULL, 0 },
    { "no-wpp",               no_argument, NULL, 0 },
    { "wpp",                  no_argument, NULL, 0 },
    { "ctu",            required_argument, NULL, 's' },
    { "tu-intra-depth", required_argument, NULL, 0 },
    { "tu-inter-depth", required_argument, NULL, 0 },
    { "me",             required_argument, NULL, 0 },
    { "subme",          required_argument, NULL, 'm' },
    { "merange",        required_argument, NULL, 0 },
    { "max-merge",      required_argument, NULL, 0 },
    { "rdpenalty",      required_argument, NULL, 0 },
    { "no-rect",              no_argument, NULL, 0 },
    { "rect",                 no_argument, NULL, 0 },
    { "no-amp",               no_argument, NULL, 0 },
    { "amp",                  no_argument, NULL, 0 },
    { "no-early-skip",        no_argument, NULL, 0 },
    { "early-skip",           no_argument, NULL, 0 },
    { "no-fast-cbf",          no_argument, NULL, 0 },
    { "fast-cbf",             no_argument, NULL, 0 },
    { "no-tskip",             no_argument, NULL, 0 },
    { "tskip",                no_argument, NULL, 0 },
    { "no-tskip-fast",        no_argument, NULL, 0 },
    { "tskip-fast",           no_argument, NULL, 0 },
    { "no-constrained-intra", no_argument, NULL, 0 },
    { "constrained-intra",    no_argument, NULL, 0 },
    { "refresh",        required_argument, NULL, 0 },
    { "keyint",         required_argument, NULL, 'i' },
    { "rc-lookahead",   required_argument, NULL, 0 },
    { "bframes",        required_argument, NULL, 'b' },
    { "bframe-bias",    required_argument, NULL, 0 },
    { "b-adapt",        required_argument, NULL, 0 },
    { "ref",            required_argument, NULL, 0 },
    { "no-weightp",           no_argument, NULL, 0 },
    { "weightp",              no_argument, NULL, 'w' },
    { "crf",            required_argument, NULL, 0 },
    { "bitrate",        required_argument, NULL, 0 },
    { "qp",             required_argument, NULL, 'q' },
    { "aq-mode",        required_argument, NULL, 0 },
    { "cbqpoffs",       required_argument, NULL, 0 },
    { "crqpoffs",       required_argument, NULL, 0 },
    { "rd",             required_argument, NULL, 0 },
    { "no-signhide",          no_argument, NULL, 0 },
    { "signhide",             no_argument, NULL, 0 },
    { "no-lft",               no_argument, NULL, 0 },
    { "lft",                  no_argument, NULL, 0 },
    { "no-sao",               no_argument, NULL, 0 },
    { "sao",                  no_argument, NULL, 0 },
    { "sao-lcu-bounds", required_argument, NULL, 0 },
    { "sao-lcu-opt",    required_argument, NULL, 0 },
    { "no-ssim",              no_argument, NULL, 0 },
    { "ssim",                 no_argument, NULL, 0 },
    { "no-psnr",              no_argument, NULL, 0 },
    { "psnr",                 no_argument, NULL, 0 },
    { "hash",           required_argument, NULL, 0 },
    { "no-strong-intra-smoothing", no_argument, NULL, 0 },
    { "strong-intra-smoothing",    no_argument, NULL, 0 },
    { 0, 0, 0, 0 }
};

struct CLIOptions
{
    Input*  input;
    Output* recon;
    std::fstream bitstreamFile;
    bool bProgress;
    bool bForceY4m;
    uint32_t totalbytes;

    uint32_t frameSkip;         // number of frames to skip from the beginning
    uint32_t framesToBeEncoded; // number of frames to encode

    int64_t startTime;
    int64_t prevUpdateTime;

    /* in microseconds */
    static const int UPDATE_INTERVAL = 250000;

    CLIOptions()
    {
        input = NULL;
        recon = NULL;
        framesToBeEncoded = frameSkip = totalbytes = 0;
        bProgress = true;
        bForceY4m = false;
        startTime = x265_mdate();
        prevUpdateTime = 0;
    }

    void destroy();
    void writeNALs(const x265_nal* nal, uint32_t nalcount);
    void printVersion(x265_param *par);
    void showHelp(x265_param *par);
    bool parse(int argc, char **argv, x265_param* par);
};

void CLIOptions::destroy()
{
    if (input)
        input->release();
    input = NULL;
    if (recon)
        recon->release();
    recon = NULL;
}

void CLIOptions::writeNALs(const x265_nal* nal, uint32_t nalcount)
{
    PPAScopeEvent(bitstream_write);
    for (uint32_t i = 0; i < nalcount; i++)
    {
        bitstreamFile.write((const char*)nal->p_payload, nal->i_payload);
        totalbytes += nal->i_payload;
        nal++;
    }
}

void CLIOptions::printVersion(x265_param *par)
{
    fprintf(stderr, "x265 [info]: HEVC encoder version %s\n", x265_version_str);
    fprintf(stderr, "x265 [info]: build info %s\n", x265_build_info_str);
    x265_setup_primitives(par, 0);
}

void CLIOptions::showHelp(x265_param *par)
{
    x265_param_default(par);
    printVersion(par);
#define H0 printf
#define OPT(value) (value ? "enabled" : "disabled")
    H0("\nSyntax: x265 [options] infile [-o] outfile\n");
    H0("    infile can be YUV or Y4M\n");
    H0("    outfile is raw HEVC bitstream\n");
    H0("\nExecutable Options:\n");
    H0("-h/--h                           Show this help text and exit\n");
    H0("-V/--version                     Show version info and exit\n");
    H0("   --cpuid                       Limit SIMD capability bitmap 0:auto 1:None. Default:0\n");
    H0("   --threads                     Number of threads for thread pool (0: detect CPU core count, default)\n");
    H0("-F/--frame-threads               Number of concurrently encoded frames. Default %d\n", par->frameNumThreads);
    H0("   --log                         Logging level 0:ERROR 1:WARNING 2:INFO 3:DEBUG -1:NONE. Default %d\n", par->logLevel);
    H0("   --csv                         Comma separated log file, log level >= 3 frame log, else one line per run\n");
    H0("   --y4m                         Parse input stream as YUV4MPEG2 regardless of file extension\n");
    H0("   --no-progress                 Disable CLI progress reports\n");
    H0("-o/--output                      Bitstream output file name\n");
    H0("\nInput Options:\n");
    H0("   --input                       Raw YUV or Y4M input file name\n");
    H0("   --input-depth                 Bit-depth of input file (YUV only) Default %d\n", par->inputBitDepth);
    H0("   --input-res                   Source picture size [w x h], auto-detected if Y4M\n");
    H0("   --input-csp                   Source color space parameter, auto-detected if Y4M\n");
    H0("   --fps                         Source frame rate, auto-detected if Y4M\n");
    H0("   --frame-skip                  Number of frames to skip at start of input file\n");
    H0("-f/--frames                      Number of frames to be encoded. Default all\n");
    H0("\nQuad-Tree analysis:\n");
    H0("   --[no-]wpp                    Enable Wavefront Parallel Processing. Default %s\n", OPT(par->bEnableWavefront));
    H0("-s/--ctu                         Maximum CU size (default: 64x64). Default %d\n", par->maxCUSize);
    H0("   --tu-intra-depth              Max TU recursive depth for intra CUs. Default %d\n", par->tuQTMaxIntraDepth);
    H0("   --tu-inter-depth              Max TU recursive depth for inter CUs. Default %d\n", par->tuQTMaxInterDepth);
    H0("\nTemporal / motion search options:\n");
    H0("   --me                          Motion search method 0:dia 1:hex 2:umh 3:star 4:full. Default %d\n", par->searchMethod);
    H0("-m/--subme                       Amount of subpel refinement to perform (0:least .. 7:most). Default %d \n", par->subpelRefine);
    H0("   --merange                     Motion search range. Default %d\n", par->searchRange);
    H0("   --[no-]rect                   Enable rectangular motion partitions Nx2N and 2NxN. Default %s\n", OPT(par->bEnableRectInter));
    H0("   --[no-]amp                    Enable asymmetric motion partitions, requires --rect. Default %s\n", OPT(par->bEnableAMP));
    H0("   --max-merge                   Maximum number of merge candidates. Default %d\n", par->maxNumMergeCand);
    H0("   --[no-]early-skip             Enable early SKIP detection. Default %s\n", OPT(par->bEnableEarlySkip));
    H0("   --[no-]fast-cbf               Enable Cbf fast mode \n \t\t\t\t Default : %s\n", OPT(par->bEnableCbfFastMode));
    H0("\nSpatial / intra options:\n");
    H0("   --rdpenalty                   penalty for 32x32 intra TU in non-I slices. 0:disabled 1:RD-penalty 2:maximum. Default %d\n", par->rdPenalty);
    H0("   --[no-]tskip                  Enable intra transform skipping. Default %s\n", OPT(par->bEnableTransformSkip));
    H0("   --[no-]tskip-fast             Enable fast intra transform skipping. Default %s\n", OPT(par->bEnableTSkipFast));
    H0("   --[no-]strong-intra-smoothing Enable strong intra smoothing for 32x32 blocks. Default %s\n", OPT(par->bEnableStrongIntraSmoothing));
    H0("   --[no-]constrained-intra      Constrained intra prediction (use only intra coded reference pixels) Default %s\n", OPT(par->bEnableConstrainedIntra));
    H0("\nSlice decision options:\n");
    H0("   --refresh                     Intra refresh type - 0:none, 1:CDR, 2:IDR (default: CDR) Default %d\n", par->decodingRefreshType);
    H0("-i/--keyint                      Max intra period in frames. Default %d\n", par->keyframeMax);
    H0("   --rc-lookahead                Number of frames for frame-type lookahead (determines encoder latency) Default %d\n", par->lookaheadDepth);
    H0("   --bframes                     Maximum number of consecutive b-frames (now it only enables B GOP structure) Default %d\n", par->bframes);
    H0("   --bframe-bias                 Bias towards B frame decisions. Default %d\n", par->bFrameBias);
    H0("   --b-adapt                     0 - none, 1 - fast, 2 - full (trellis) adaptive B frame scheduling. Default %d\n", par->bFrameAdaptive);
    H0("   --ref                         max number of L0 references to be allowed (1 .. 16) Default %d\n", par->maxNumReferences);
    H0("-w/--[no-]weightp                Enable weighted prediction in P slices. Default %s\n", OPT(par->bEnableWeightedPred));
    H0("\nQP, rate control and rate distortion options:\n");
    H0("   --bitrate                     Target bitrate (kbps), implies ABR. Default %d\n", par->rc.bitrate);
    H0("-q/--qp                          Base QP for CQP mode. Default %d\n", par->rc.qp);
    H0("   --cbqpoffs                    Chroma Cb QP Offset. Default %d\n", par->cbQpOffset);
    H0("   --crqpoffs                    Chroma Cr QP Offset. Default %d\n", par->crQpOffset);
    H0("   --rd                          Level of RD in mode decision 0:least....2:full RDO. Default %d\n", par->rdLevel);
    H0("   --[no-]signhide               Hide sign bit of one coeff per TU (rdo). Default %s\n", OPT(par->bEnableSignHiding));
    H0("\nLoop filter:\n");
    H0("   --[no-]lft                    Enable Loop Filter. Default %s\n", OPT(par->bEnableLoopFilter));
    H0("\nSample Adaptive Offset loop filter:\n");
    H0("   --[no-]sao                    Enable Sample Adaptive Offset. Default %s\n", OPT(par->bEnableSAO));
    H0("   --sao-lcu-bounds              0: right/bottom boundary areas skipped  1: non-deblocked pixels are used. Default %d\n", par->saoLcuBoundary);
    H0("   --sao-lcu-opt                 0: SAO picture-based optimization, 1: SAO LCU-based optimization. Default %d\n", par->saoLcuBasedOptimization);
    H0("\nQuality reporting metrics:\n");
    H0("   --[no-]ssim                   Enable reporting SSIM metric scores. Default %s\n", OPT(par->bEnableSsim));
    H0("   --[no-]psnr                   Enable reporting PSNR metric scores. Default %s\n", OPT(par->bEnablePsnr));
    H0("\nReconstructed video options (debugging):\n");
    H0("-r/--recon                       Reconstructed raw image YUV or Y4M output file name\n");
    H0("   --recon-depth                 Bit-depth of reconstructed raw image file. Default 8\n");
    H0("\nSEI options:\n");
    H0("   --hash                        Decoded Picture Hash SEI 0: disabled, 1: MD5, 2: CRC, 3: Checksum. Default %d\n", par->decodedPictureHashSEI);
#undef OPT
#undef H0
    exit(0);
}

bool CLIOptions::parse(int argc, char **argv, x265_param* par)
{
    int berror = 0;
    int help = 0;
    int cpuid = 0;
    int reconFileBitDepth = 8;
    const char *inputfn = NULL;
    const char *reconfn = NULL;
    const char *bitstreamfn = NULL;
    const char *inputRes = NULL;
    const char *preset = "medium";
    const char *tune = "psnr";

    /* Presets are applied before all other options. */
    for (optind = 0;; )
    {
        int c = getopt_long(argc, argv, short_options, long_options, NULL);
        if (c == -1)
            break;
        if (c == 'p')
            preset = optarg;
        if (c == 't')
            tune = optarg;
        else if (c == '?')
            return true;
    }

    if (x265_param_default_preset(par, preset, tune) < 0)
        return true;

    for (optind = 0;; )
    {
        int long_options_index = -1;
        int c = getopt_long(argc, argv, short_options, long_options, &long_options_index);
        if (c == -1)
        {
            break;
        }

        switch (c)
        {
        case 'h':
            showHelp(par);
            break;

        case 'V':
            printVersion(par);
            exit(0);

        default:
            if (long_options_index < 0 && c > 0)
            {
                for (size_t i = 0; i < sizeof(long_options) / sizeof(long_options[0]); i++)
                {
                    if (long_options[i].val == c)
                    {
                        long_options_index = (int)i;
                        break;
                    }
                }

                if (long_options_index < 0)
                {
                    /* getopt_long might have already printed an error message */
                    if (c != 63)
                        x265_log(NULL, X265_LOG_WARNING, "internal error: short option '%c' has no long option\n", c);
                    return true;
                }
            }
            if (long_options_index < 0)
            {
                x265_log(NULL, X265_LOG_WARNING, "short option '%c' unrecognized\n", c);
                return true;
            }
#define OPT(longname) \
    else if (!strcmp(long_options[long_options_index].name, longname))

            if (0) ;
            OPT("cpuid") cpuid = atoi(optarg);
            OPT("frames") this->framesToBeEncoded = (uint32_t)atoi(optarg);
            OPT("preset") preset = optarg;
            OPT("tune") tune = optarg;
            OPT("no-progress") this->bProgress = false;
            OPT("frame-skip") this->frameSkip = (uint32_t)atoi(optarg);
            OPT("output") bitstreamfn = optarg;
            OPT("input") inputfn = optarg;
            OPT("recon") reconfn = optarg;
            OPT("input-depth") par->inputBitDepth = (uint32_t)atoi(optarg);
            OPT("recon-depth") reconFileBitDepth = (uint32_t)atoi(optarg);
            OPT("input-res") inputRes = optarg;
            OPT("y4m") bForceY4m = true;
            else
                berror |= x265_param_parse(par, long_options[long_options_index].name, optarg);

            if (berror)
            {
                const char *name = long_options_index > 0 ? long_options[long_options_index].name : argv[optind - 2];
                x265_log(NULL, X265_LOG_ERROR, "invalid argument: %s = %s\n", name, optarg);
                return true;
            }
#undef OPT
        }
    }

    if (optind < argc && !inputfn)
        inputfn = argv[optind++];
    if (optind < argc && !bitstreamfn)
        bitstreamfn = argv[optind++];
    if (optind < argc)
    {
        x265_log(par, X265_LOG_WARNING, "extra unused command arguments given <%s>\n", argv[optind]);
        return true;
    }

    if (argc <= 1 || help)
        showHelp(par);

    if (inputfn == NULL || bitstreamfn == NULL)
    {
        x265_log(par, X265_LOG_ERROR, "input or output file not specified, try -V for help\n");
        return true;
    }
    this->input = Input::open(inputfn, par->inputBitDepth, bForceY4m);
    if (!this->input || this->input->isFail())
    {
        x265_log(par, X265_LOG_ERROR, "unable to open input file <%s>\n", inputfn);
        return true;
    }
    if (this->input->getWidth())
    {
        /* parse the width, height, frame rate from the y4m file */
        par->sourceCsp = this->input->getColorSpace();
        par->sourceWidth = this->input->getWidth();
        par->sourceHeight = this->input->getHeight();
        par->frameRate = (int)this->input->getRate();
    }
    else if (inputRes)
    {
        this->input->setColorSpace(par->sourceCsp);
        sscanf(inputRes, "%dx%d", &par->sourceWidth, &par->sourceHeight);
        this->input->setDimensions(par->sourceWidth, par->sourceHeight);
        this->input->setBitDepth(par->inputBitDepth);
    }
    else if (par->sourceHeight <= 0 || par->sourceWidth <= 0 || par->frameRate <= 0)
    {
        x265_log(par, X265_LOG_ERROR, "YUV input requires source width, height, and rate to be specified\n");
        return true;
    }
    else
    {
        this->input->setDimensions(par->sourceWidth, par->sourceHeight);
        this->input->setBitDepth(par->inputBitDepth);
    }

    if (reconFileBitDepth > 0)
    {
        if (reconFileBitDepth != par->inputBitDepth)
        {
            x265_log(par, X265_LOG_ERROR, "Bit depth of the recon file should be the same as input bit depth\n");
            /* TODO: Support recon files with bitdepth > input bit depth??*/
            return true;
        }
    }
    else
    {
        reconFileBitDepth = par->inputBitDepth;
    }

    int guess = this->input->guessFrameCount();
    if (this->frameSkip)
    {
        this->input->skipFrames(this->frameSkip);
    }

    uint32_t fileFrameCount = guess < 0 ? 0 : (uint32_t)guess;
    if (this->framesToBeEncoded && fileFrameCount)
        this->framesToBeEncoded = X265_MIN(this->framesToBeEncoded, fileFrameCount - this->frameSkip);
    else if (fileFrameCount)
        this->framesToBeEncoded = fileFrameCount - this->frameSkip;

    if (par->logLevel >= X265_LOG_INFO)
    {
        if (this->framesToBeEncoded == 0)
            fprintf(stderr, "%s  [info]: %dx%d %dHz %s, unknown frame count\n", input->getName(),
                    par->sourceWidth, par->sourceHeight, par->frameRate,
                    (par->sourceCsp >= X265_CSP_I444) ? "C444" : (par->sourceCsp >= X265_CSP_I422) ? "C422" : "C420");
        else
            fprintf(stderr, "%s  [info]: %dx%d %dHz %s, frames %u - %d of %d\n", input->getName(),
                    par->sourceWidth, par->sourceHeight, par->frameRate,
                    (par->sourceCsp >= X265_CSP_I444) ? "C444" : (par->sourceCsp >= X265_CSP_I422) ? "C422" : "C420",
                    this->frameSkip, this->frameSkip + this->framesToBeEncoded - 1, fileFrameCount);
    }

    this->input->startReader();

    if (reconfn)
    {
        this->recon = Output::open(reconfn, par->sourceWidth, par->sourceHeight, reconFileBitDepth, par->frameRate, par->sourceCsp);
        if (this->recon->isFail())
        {
            x265_log(par, X265_LOG_WARNING, "unable to write reconstruction file\n");
            this->recon->release();
            this->recon = 0;
        }
    }

#if HIGH_BIT_DEPTH
    if (par->inputBitDepth != 12 && par->inputBitDepth != 10 && par->inputBitDepth != 8)
    {
        x265_log(par, X265_LOG_ERROR, "Only bit depths of 8, 10, or 12 are supported\n");
        return true;
    }
#else
    if (par->inputBitDepth != 8)
    {
        x265_log(par, X265_LOG_ERROR, "not compiled for bit depths greater than 8\n");
        return true;
    }
#endif // if HIGH_BIT_DEPTH

    this->bitstreamFile.open(bitstreamfn, std::fstream::binary | std::fstream::out);
    if (!this->bitstreamFile)
    {
        x265_log(NULL, X265_LOG_ERROR, "failed to open bitstream file <%s> for writing\n", bitstreamfn);
        return true;
    }

    x265_setup_primitives(par, cpuid);
    printVersion(par);
    return false;
}

int main(int argc, char *argv[])
{
    CLIOptions   cliopt;

    if (cliopt.parse(argc, argv, &param))
    {
        cliopt.destroy();
        exit(1);
    }

    param.bEnableStrongIntraSmoothing = false; //No strong intra smoothing for competition

    x265_encoder *encoder = x265_encoder_open(&param);
    if (!encoder)
    {
        x265_log(&param, X265_LOG_ERROR, "failed to open encoder\n");
        cliopt.destroy();
        x265_cleanup();
        exit(1);
    }

	Encoder *encoder_c = static_cast<Encoder*>(encoder);

    //Initialize arrays for storing neighboring pixel values
    refAbove1 = (Pel*)X265_MALLOC(Pel, 3 * MAX_CU_SIZE);
    refAbove2 = (Pel*)X265_MALLOC(Pel, 3 * MAX_CU_SIZE);
    refLeft1 = (Pel*)X265_MALLOC(Pel, 3 * MAX_CU_SIZE);
    refLeft2 = (Pel*)X265_MALLOC(Pel, 3 * MAX_CU_SIZE);
    predBufHeight  = ((MAX_CU_SIZE + 2) << 4);
    predBufStride = ((MAX_CU_SIZE  + 8) << 4);
    predBuf = new Pel[predBufStride * predBufHeight];

    //Save globals so we can restore them at the end
    //We need to restore the original values before destroy()ing data structures because many of the destroy() functions
    //use these globals to determine the size of their arrays
    int g_maxCUDepth_bak = g_maxCUDepth;
    int g_addCUDepth_bak = g_addCUDepth;
    int g_maxCUWidth_bak = g_maxCUWidth;
    int g_maxCUHeight_bak = g_maxCUHeight;

    g_maxCUDepth = 0; //Disallow recursion to decompose frames into a regular grid of equal size CUs.
    g_addCUDepth = 0;
    //NOTE: has to be after x265_encoder_open() call, since that calls x265_set_globals(), which resets g_maxCUDepth.
    x265_picture pic_orig;
    x265_picture *pic_in = &pic_orig;

    x265_picture_init(&param, pic_in);

    uint32_t inFrameCount = 0;

    TComPic *pic = NULL;

    while (1)
    {
        pic_orig.poc = inFrameCount;
        if (cliopt.framesToBeEncoded && inFrameCount >= cliopt.framesToBeEncoded)
            break;
        else if (cliopt.input->readPicture(pic_orig))
            inFrameCount++;
        else
            break;

        ece408_intra_pred_result *ref = ece408_competition_ref(encoder_c, pic_in, pic, 1);
        
        ece408_frame frame(param.sourceWidth, param.sourceHeight, pic_in);

        //Uncomment this one to run the student version
        ece408_intra_pred_result *student = ece408_competition(&frame, 1);
        //Uncomment this one instead to run the reference version twice (to test the compare function)
        //ece408_intra_pred_result *student = ece408_competition_ref(encoder_c, pic_in, pic, 1);

#ifdef VERIFY
        if(!ece408_compare(ref, student, 1)) {
        	printf("Error in frame %d\n", inFrameCount);
        	exit(1);
        }
#endif
        for(int i = 0; i < 4*1; i++) {
        	ref[i].destroy();
            student[i].destroy();
        }
        delete[] ref;
        delete[] student;
    }
    
#ifdef VERIFY
    printf("Success!\n");
#endif

    //Restore globals
    g_maxCUDepth = g_maxCUDepth_bak;
    g_addCUDepth = g_addCUDepth_bak;
    g_maxCUWidth = g_maxCUWidth_bak;
    g_maxCUHeight = g_maxCUHeight_bak;

    pic->destroy(param.bframes);
    delete pic;

    x265_encoder_close(encoder);

    X265_FREE(refAbove1);
    X265_FREE(refAbove2);
    X265_FREE(refLeft1);
    X265_FREE(refLeft2);
    delete[] predBuf;

    orig_yuv.destroy();
    pred_yuv.destroy();

    x265_cleanup(); /* Free library singletons */
    cliopt.destroy();

    return 0;
}

void ece408_setup_pic(x265_encoder *enc, TComPic* &pic_ptr, int cu_size) {
    Encoder *encoder = static_cast<Encoder*>(enc);

    g_maxCUHeight = cu_size;
    g_maxCUWidth = cu_size;
    #ifdef VERBOSE
    printf("%u x %u CU size\n", g_maxCUHeight, g_maxCUWidth);
    #endif
    if(!pic_ptr) {
        pic_ptr = new TComPic;
        pic_ptr->create(encoder);
    }
    else {
        pic_ptr->destroy(param.bframes);
	    pic_ptr->create(encoder);
    }

    // initialize SPS
    encoder->initSPS(&sps);
    pps.setSPS(&sps);
    encoder->initPPS(&pps);
    TComSlice* slice = pic_ptr->getSlice();
    slice->setSPS(&sps);
    slice->setPPS(&pps);
    slice->setSliceBits(0);
    slice->setPic(pic_ptr);
    slice->initSlice();

	#ifdef VERBOSE     
    printf("%u x %u CUs, %u CUs in frame\n", pic_ptr->getFrameWidthInCU(), pic_ptr->getFrameHeightInCU(), pic_ptr->getNumCUsInFrame());
    #endif
}

void ece408_load_pic(x265_encoder *enc, x265_picture *pic_in, TComPic *pic) {
    Encoder *encoder = static_cast<Encoder*>(enc);
    pic->getPicYuvOrg()->copyFromPicture(*pic_in, encoder->m_pad);
    //Need to copy the original picture to the reconstructed member too because TComPattern gets its neighbor samples from there.
    //TODO can we just make the Rec member point at the Org member and avoid 2 copies?
    pic->getPicYuvRec()->copyFromPicture(*pic_in, encoder->m_pad);
}

//channel = 0 for luma, 1 for cb, 2 for cr
void ece408_intra_pred_channel(TComPic *pic, TComDataCU *cu, int luma_size, int channel, int32_t *sad_ptr) {
//#define VERBOSE
#ifdef VERBOSE
	int cuAddr = cu->getAddr();
   	printf("refAbove1: ");
   	for(int i = 0; i < 64*3; i++)
   		printf("%d ", refAbove1[i]);
   	printf("\n");
   	printf("refAbove2: ");
   	for(int i = 0; i < 64*3; i++)
   		printf("%d ", refAbove2[i]);
   	printf("\n");
   	printf("refLeft1: ");
   	for(int i = 0; i < 64*3; i++)
   		printf("%d ", refLeft1[i]);
   	printf("\n");
   	printf("refLeft2: ");
   	for(int i = 0; i < 64*3; i++)
   		printf("%d ", refLeft2[i]);
   	printf("\n");
#endif
	int chroma_size = luma_size >> 1;
	bool luma = (channel == 0);
	bool cb = (channel == 1);
	bool cr = (channel == 2);
	int size = luma ? luma_size : chroma_size;
	Pel* orig_pel   = luma ? orig_yuv.getLumaAddr(0, size) : (cb ? orig_yuv.getCbAddr(0, size) : orig_yuv.getCrAddr(0, size));
    Pel* pred_pel   = luma ? pred_yuv.getLumaAddr(0, size) : (cb ? pred_yuv.getCbAddr(0, size) : pred_yuv.getCrAddr(0, size));
    uint32_t stride = luma ? pred_yuv.getStride() : pred_yuv.getCStride();

    Pel *pAboveUnfilt = (cr ? refAbove2 : refAbove1) + size - 1;
    Pel *pAboveFilt = luma ? (refAbove2 + size - 1) : pAboveUnfilt;
    Pel *pLeftUnfilt = (cr ? refLeft2 : refLeft1) + size - 1;
    Pel *pLeftFilt = luma ? (refLeft2  + size - 1) : pLeftUnfilt;

    int nLog2SizeMinus2 = g_convertToBit[size];
    pixelcmp_t sa8d = primitives.sa8d[nLog2SizeMinus2];

    #ifdef VERBOSE
    printf("CU %u channel %d Orig:\n", cuAddr, channel);
    for(int row = 0; row < size; row++) {
        for(int col = 0; col < size; col++) {
            printf("%02X ", orig_pel[row*size + col]);
        }
        printf("\n");
    }
    #endif

    int sad;

    Pel *above = (luma && size >= 8) ? pAboveFilt : pAboveUnfilt;
    Pel *left  = (luma && size >= 8) ? pLeftFilt : pLeftUnfilt;

    //TODO check to make sure we're filtering in all the right conditions
    primitives.intra_pred_planar(above + 1, left + 1, pred_pel, stride, size);
    sad = sa8d(orig_pel, stride, pred_pel, stride);
    *(sad_ptr++) = sad;
    #ifdef VERBOSE
    printf("CU %u channel %d, planar SAD = %d\n", cuAddr, channel, sad);
    #endif

    //TODO check to make sure we're filtering in all the right conditions
    primitives.intra_pred_dc(pAboveUnfilt + 1, pLeftUnfilt + 1, pred_pel, stride, size, (size <= 16));
    sad = sa8d(orig_pel, stride, pred_pel, stride);
    *(sad_ptr++) = sad;
    #ifdef VERBOSE
    printf("CU %u channel %d, size = %d, stride = %d, DC:\n", cuAddr, channel, size, stride);
    for(int row = 0; row < size; row++) {
    	for(int col = 0; col < size; col++) {
    		printf("%02X ", pred_pel[row*size+col]);
    	}
    	printf("\n");
    }
    printf("SAD = %d\n", sad);
    #endif

    primitives.transpose[nLog2SizeMinus2](buf_trans, orig_pel, stride);
    //TODO check to make sure we're filtering in all the right conditions
    primitives.intra_pred_allangs[nLog2SizeMinus2](tmp, pAboveUnfilt, pLeftUnfilt, pAboveFilt, pLeftFilt, (luma && (size <= 16)));
    #ifdef VERBOSE
    printf("CU %u channel %d, angular SAD = ", cuAddr, channel);
    #endif
    for (int mode = 2; mode < 35; mode++)
    {
        bool modeHor = (mode < 18);
        Pel *cmp = (modeHor ? buf_trans : orig_pel);
        intptr_t srcStride = (modeHor ? size : stride);
    #ifdef VERBOSE
    	printf("Pred mode %d\n", mode);
    	for(int r = 0; r < size; r++) {
    		for(int c = 0; c < size; c++)
    			printf("%02X ", tmp[(mode-2) * (size * size) + r * size + c]);
    		printf("\n");
    	}
   	#endif
        sad = sa8d(cmp, srcStride, &tmp[(mode - 2) * (size * size)], size);
        *(sad_ptr++) = sad;
    #ifdef VERBOSE
        printf("%d, ", sad);
    #endif
    }
    #ifdef VERBOSE
    printf("\n");
    #endif
}
//#undef VERBOSE

// TComPattern doesn't have a version for chroma that creates buffered references
// NOTE: No filtering of neighbor samples is done for chroma
void initAdiPatternChroma(TComPattern *pat, TComDataCU* cu, uint32_t zOrderIdxInPart, uint32_t partDepth, Pel* adiBuf, int strideOrig, int heightOrig,
                                 Pel* refAboveCb, Pel* refLeftCb, Pel* refAboveCr, Pel* refLeftCr)
{
    pat->initAdiPatternChroma(cu, zOrderIdxInPart, partDepth, adiBuf, strideOrig, heightOrig);
    uint32_t cuWidth   = cu->getWidth(0) >> partDepth >> 1; //>> 1 to account for 4:2:0 subsampling
    uint32_t cuHeight  = cu->getHeight(0) >> partDepth >> 1;
    uint32_t cuWidth2  = cuWidth << 1;
    uint32_t cuHeight2 = cuHeight << 1;

    refAboveCb += cuWidth - 1;
    refLeftCb += cuWidth - 1;

    refAboveCr += cuWidth - 1;
    refLeftCr += cuWidth - 1;

    memcpy(refAboveCb, adiBuf, (cuWidth2 + 1) * sizeof(Pel));
    memcpy(refAboveCr, adiBuf + ADI_BUF_STRIDE * (2 * cuHeight + 1), (cuWidth2 + 1) * sizeof(Pel));

    for (unsigned int k = 0; k < cuHeight2 + 1; k++)
    {
        refLeftCb[k] = adiBuf[k * ADI_BUF_STRIDE];
        refLeftCr[k] = (adiBuf + ADI_BUF_STRIDE * (cuHeight2 + 1))[k * ADI_BUF_STRIDE];
    }
}

//luma_size is the (square) block size of luma blocks, chroma blocks are assumed (luma_size/2)x(luma_size/2)
void ece408_intra_pred(TComPic *pic, TComDataCU *cu, int luma_size, int32_t *y_ptr, int32_t *cb_ptr, int32_t *cr_ptr) {
	int cuAddr = cu->getAddr();
    orig_yuv.copyFromPicLuma(pic->getPicYuvOrg(), cuAddr, 0);
    if(luma_size > 4)
        orig_yuv.copyFromPicChroma(pic->getPicYuvOrg(), cuAddr, 0);

	cu->getPattern()->initPattern(cu, /*initTrDepth*/ 0, /*partOffset*/ 0);
    cu->getPattern()->initAdiPattern(cu, /*partOffset*/ 0, /*initTrDepth*/ 0, predBuf, predBufStride, predBufHeight, refAbove1, refLeft1, refAbove2, refLeft2);

	ece408_intra_pred_channel(pic, cu, luma_size, 0, y_ptr);

	if(luma_size > 4) { //No 2x2 chroma blocks, and 4x4 chroma blocks are covered with 8x8 luma
		initAdiPatternChroma(cu->getPattern(), cu, 0, 0, predBuf, predBufStride, predBufHeight, refAbove1, refLeft1, refAbove2, refLeft2);
		ece408_intra_pred_channel(pic, cu, luma_size, 1, cb_ptr);
		ece408_intra_pred_channel(pic, cu, luma_size, 2, cr_ptr);
	}
}

ece408_intra_pred_result *ece408_competition_ref(Encoder *encoder, x265_picture *pics_in, TComPic* &pic, int num_frames) { 
	ece408_intra_pred_result *ret = new ece408_intra_pred_result[4*num_frames]; //8x8,16x16,32x32,64x64
	ece408_intra_pred_result *cur_result = ret;

	for(int i = 0; i < num_frames; i++) {
		for(int luma_size_shift = 2; luma_size_shift <= 5; luma_size_shift++) {
	        int luma_size = 1 << luma_size_shift; // luma_size x luma_size luma PBs
	        cur_result->create(param.sourceWidth, param.sourceHeight, luma_size);

	        int32_t *y_satd_results = cur_result->y_satd_results;
		    uint8_t *y_modes = cur_result->y_modes;
		    int32_t *cb_satd_results = cur_result->cb_satd_results;
		    uint8_t *cb_modes = cur_result->cb_modes;
		    int32_t *cr_satd_results = cur_result->cr_satd_results;
		    uint8_t *cr_modes = cur_result->cr_modes;

	        ece408_setup_pic(encoder, pic, luma_size);
	        ece408_load_pic(encoder, &pics_in[i], pic);

	        //TODO would like not to have to destroy and rebuild the entire TComPic for different CU sizes, since the underlying data arrays are the same.
	        orig_yuv.destroy();
	        orig_yuv.create(luma_size, luma_size, CHROMA_420);
	        pred_yuv.destroy();
	        pred_yuv.create(luma_size, luma_size, CHROMA_420);

            //cuIndex != cuAddr because pic includes partial CUs that fall off the right and bottom edges of the frame.
            //In our case, since we're not subdividing the CUs at all, partial CUs are meaningless (they are never actually coded in an HEVC bitstream),
            //so we only care about full CUs.
            for(unsigned int cuIndex = 0; cuIndex < (unsigned int)((param.sourceWidth/luma_size)*(param.sourceHeight/luma_size)); cuIndex++) {
                unsigned int cuAddr = ((cuIndex / (param.sourceWidth/luma_size))*(pic->getFrameWidthInCU())) + (cuIndex % (param.sourceWidth/luma_size)); //TODO structure for better CSE
	            TComDataCU* cu = pic->getCU(cuAddr);
	            cu->initCU(pic, cuAddr);

	            ece408_intra_pred(pic,
	            	              cu,
	            	              luma_size,
	            	              &(y_satd_results[35*cuIndex]),
	            	              &(cb_satd_results[35*cuIndex]),
	            	              &(cr_satd_results[35*cuIndex]));
	            //printf("SATD results: ");
	            //for(int l = 0; l < 35; l++) {
	            //	printf("(%d, %d, %d, %d) ", l, y_satd_results[35*cuIndex+l], cb_satd_results[35*cuIndex+l], cr_satd_results[35*cuIndex+l]);
	            //}
	            //printf("\n");
	            for(int mode = 0; mode < 35; mode++) {
	            	y_satd_results[35*cuIndex + mode] = (y_satd_results[35*cuIndex + mode] << 8) | mode;
                    if(luma_size > 4) {
	            	  cb_satd_results[35*cuIndex + mode] = (cb_satd_results[35*cuIndex + mode] << 8) | mode;
	            	  cr_satd_results[35*cuIndex + mode] = (cr_satd_results[35*cuIndex + mode] << 8) | mode;
                    }
	            }
	            std::sort(&(y_satd_results[35*cuIndex]), &(y_satd_results[35*cuIndex+35]));
                if(luma_size > 4) {
	               std::sort(&(cb_satd_results[35*cuIndex]), &(cb_satd_results[35*cuIndex+35]));
	               std::sort(&(cr_satd_results[35*cuIndex]), &(cr_satd_results[35*cuIndex+35]));
                }
	            for(int mode = 0; mode < 35; mode++) {
	            	y_modes[35*cuIndex+mode] = (y_satd_results[35*cuIndex+mode] & 0xFF);
	            	y_satd_results[35*cuIndex+mode] >>= 8;
                    if(luma_size > 4) {
	            	  cb_modes[35*cuIndex+mode] = (cb_satd_results[35*cuIndex+mode] & 0xFF);
	            	  cb_satd_results[35*cuIndex+mode] >>= 8;
	            	  cr_modes[35*cuIndex+mode] = (cr_satd_results[35*cuIndex+mode] & 0xFF);
	            	  cr_satd_results[35*cuIndex+mode] >>= 8;
                    }
	            }
	        }
#ifdef MODE_HIST
	        int ymode_hist[35], cbmode_hist[35], crmode_hist[35];
	        for(int l = 0; l < 35; l++) {
	        	ymode_hist[l] = cbmode_hist[l] = crmode_hist[l] = 0;
	        }
	        for(int l = 0; l < (35*((param.sourceWidth/luma_size)*(param.sourceHeight/luma_size))); l += 1) { //Change to += 35 for histogram of best modes
	        	ymode_hist[y_modes[l]]++;
                if(luma_size > 4) {
	        	  cbmode_hist[cb_modes[l]]++;
	        	  crmode_hist[cr_modes[l]]++;
                }
	        }
	        printf("ymode hist: ");
	        for(int l = 0; l < 35; l++)
	        	printf("%d ", ymode_hist[l]);
            if(luma_size > 4) {
    	        printf("\ncbmode hist: ");
    	        for(int l = 0; l < 35; l++)
    	        	printf("%d ", cbmode_hist[l]);
    	        printf("\ncrmode hist: ");
    	        for(int l = 0; l < 35; l++)
    	        	printf("%d ", crmode_hist[l]);
            }
	        printf("\n");
#endif
	        cur_result++;
	    }
	}
	return ret;
}

//TODO sort student results by satd result *and* mode number to make sure we have *exactly* the same bytes in
//both arrays, even if several modes have the same SATD value.
//We want to do the sort here so that students are not required to (it's not necessary in a real x265 use case).
bool ece408_compare(ece408_intra_pred_result *ref, ece408_intra_pred_result *student, int num_frames) {
	if(student == NULL) {
		printf("Student result array pointer is NULL\n");
		return false;
	}
	for(int i = 0; i < (4*num_frames); i++) {
		if(ref[i].luma_block_size != student[i].luma_block_size) {
			printf("Ref result %d luma block size = %d, student = %d\n", i, ref[i].luma_block_size, student[i].luma_block_size);
			return false;
		}
		if(ref[i].num_blocks != student[i].num_blocks) {
			printf("Ref result %d num_blocks = %d, student = %d\n", i, ref[i].num_blocks, student[i].num_blocks);
			return false;
		}
		if(memcmp(ref[i].y_modes, student[i].y_modes, 35*ref[i].num_blocks*sizeof(*ref[i].y_modes))) {
			printf("Result %d, ref and student y_modes mismatched\n", i);
			return false;
		}
		if(memcmp(ref[i].y_satd_results, student[i].y_satd_results, 35*ref[i].num_blocks*sizeof(*ref[i].y_satd_results))) {
			printf("Result %d, ref and student y_satd_results mismatched\n", i);
			return false;
		}
		if(ref[i].luma_block_size > 4) {
			if(memcmp(ref[i].cb_modes, student[i].cb_modes, 35*ref[i].num_blocks*sizeof(*ref[i].cb_modes))) {
				printf("Result %d, ref and student cb_modes mismatched\n", i);
				return false;
			}
			if(memcmp(ref[i].cb_satd_results, student[i].cb_satd_results, 35*ref[i].num_blocks*sizeof(*ref[i].cb_satd_results))) {
				printf("Result %d, ref and student cb_satd_results mismatched\n", i);
				return false;
			}
			if(memcmp(ref[i].cr_modes, student[i].cr_modes, 35*ref[i].num_blocks*sizeof(*ref[i].cr_modes))) {
				printf("Result %d, ref and student cr_modes mismatched\n", i);
				return false;
			}
			if(memcmp(ref[i].cr_satd_results, student[i].cr_satd_results, 35*ref[i].num_blocks*sizeof(*ref[i].cr_satd_results))) {
				printf("Result %d, ref and student cr_satd_results mismatched\n", i);
				return false;
			}
		}
	}
	return true;
}

ece408_intra_pred_result *ece408_competition(ece408_frame *imgs, int num_frames) {
	//Fill in your own!
	(void)imgs;
	ece408_intra_pred_result *ret = new ece408_intra_pred_result[4*num_frames]; //8x8,16x16,32x32,64x64
	ece408_intra_pred_result *cur_result = ret;

        hipError_t cuda_ret;

        uint8_t *d_y,
                *d_cr,
                *d_cb;

        unsigned int y_size = ((imgs->width) * (imgs->height));
        unsigned int cr_size,
                     cb_size;

        // TO DO : do we need a ceil here ? 
        cr_size = cb_size = (y_size/2);

        // Allocate global memorcy for y, cr, cb components of the frame
        cuda_ret = hipMalloc((void **) &d_y, y_size);
        if ( cuda_ret != hipSuccess )
        {
            printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        cuda_ret = hipMalloc((void **) &d_cr, cr_size);
        if ( cuda_ret != hipSuccess )
        {
            printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        cuda_ret = hipMalloc((void **) &d_cb, cb_size);
        if ( cuda_ret != hipSuccess )
        {
            printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        cuda_ret = hipDeviceSynchronize();
        if ( cuda_ret != hipSuccess )
        {
            printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        cuda_ret = hipMemcpy(d_y, imgs->y, y_size, hipMemcpyHostToDevice);
        if ( cuda_ret != hipSuccess )
        {
            printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        cuda_ret = hipMemcpy(d_cr, imgs->cr, cr_size, hipMemcpyHostToDevice);
        if ( cuda_ret != hipSuccess )
        {
            printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        cuda_ret = hipMemcpy(d_cb, imgs->cb, cb_size, hipMemcpyHostToDevice);
        if ( cuda_ret != hipSuccess )
        {
            printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

        cuda_ret = hipDeviceSynchronize();
        if ( cuda_ret != hipSuccess )
        {
            printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
            exit(EXIT_FAILURE);
        }

	for(int i = 0; i < num_frames; i++) {
		for(int luma_size_shift = 2; luma_size_shift <= 5; luma_size_shift++) {
	        int luma_size = 1 << luma_size_shift; // luma_size x luma_size luma PBs
	        cur_result->create(32, 32, luma_size);

                // Start
 
                int32_t *d_res_y;
                int32_t *d_res_cr;
                int32_t *d_res_cb;
                uint8_t *d_y_modes;
                uint8_t *d_cr_modes;
                uint8_t *d_cb_modes;

                unsigned int y_res_size = (35 * (cur_result->num_blocks)); 
                unsigned int cr_res_size,
                             cb_res_size;

                cr_res_size = cb_res_size = y_res_size;

                // Allocate result in the device 
                cuda_ret = hipMalloc((void **) &d_res_y, y_res_size);
                if ( cuda_ret != hipSuccess )
                {
                    printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
                    exit(EXIT_FAILURE);
                }

                cuda_ret = hipMalloc((void **) &d_y_modes, y_res_size);
                if ( cuda_ret != hipSuccess )
                {
                    printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
                    exit(EXIT_FAILURE);
                }

                if ( luma_size > 4 )
                {
                    cuda_ret = hipMalloc((void **) &d_res_cr, cr_res_size);
                    if ( cuda_ret != hipSuccess )
                    {
                        printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
                        exit(EXIT_FAILURE);
                    }

                    cuda_ret = hipMalloc((void **) &d_res_cb, cb_res_size);
                    if ( cuda_ret != hipSuccess )
                    {
                        printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
                        exit(EXIT_FAILURE);
                    }

                    cuda_ret = hipMalloc((void **) &d_cr_modes, cr_res_size);
                    if ( cuda_ret != hipSuccess )
                    {
                        printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
                        exit(EXIT_FAILURE);
                    }

                    cuda_ret = hipMalloc((void **) &d_cb_modes, cb_res_size);
                    if ( cuda_ret != hipSuccess )
                    {
                        printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
                        exit(EXIT_FAILURE);
                    }
                }

                cuda_ret = hipDeviceSynchronize();
                if ( cuda_ret != hipSuccess )
                {
                    printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
                    exit(EXIT_FAILURE);
                }

                // Grid dimension
                dim3 dimGrid = dim3((int)ceil((imgs->width)/(float)luma_size), (int)ceil((imgs->height)/(float)luma_size), 1);

                // Block dimension
                dim3 dimBlock = dim3(luma_size, luma_size, 1);

                int neighbour_array_size = luma_size*2+1;
                hevcPredictionKernel<<<dimGrid, dimBlock, (neighbour_array_size * sizeof(uint8_t))>>>(d_y, d_cr, d_cb, d_res_y, d_res_cr, d_res_cb, d_y_modes, d_cr_modes, d_cb_modes, imgs->height, imgs->width);

                cuda_ret = hipDeviceSynchronize();
                if ( cuda_ret != hipSuccess )
                {
                    printf("\n%s in %s at line %d\n", hipGetErrorString(cuda_ret), __FILE__, __LINE__);
                    exit(EXIT_FAILURE);
                }

	        cur_result++;
	    }
	}
	return ret;
}
