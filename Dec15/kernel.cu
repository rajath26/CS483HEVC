#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////
//****************************************************************************
//
//    FILE NAME: kernel.cu
//
//    DECSRIPTION: This is the source file containing the kernel 
//                 for the HEVC encoding  
//
//    OPERATING SYSTEM: Linux UNIX only
//    TESTED ON:
//
//    CHANGE ACTIVITY:
//    Date        Who      Description
//    ==========  =======  ===============
//    12-11-2013	   Initial creation
//
//****************************************************************************
//////////////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include <math.h>

#define ZERO 0 
#define ONE 1
#define TWO 2
#define THREE 3
#define MINUS -1
#define DC_MODE 1 
#define PLANAR_MODE 0 
#define BITDEPTHY 8
#define BITDEPTHC 8
#define ANGULAR_18 18
#define ANGULAR_26 26
#define  ANGULAR_10 10
#define TOTAL_MODES 35
#define MAX_BLOCK_SIZE 32
#define IA_MODES 16
#define BITS_PER_SUM (8 * sizeof(sum_t))

#define HADAMARD4(d0, d1, d2, d3, s0, s1, s2, s3) { \
         sum2_t t0 = s0 + s1; \
         sum2_t t1 = s0 - s1; \
         sum2_t t2 = s2 + s3; \
         sum2_t t3 = s2 - s3; \
         d0 = t0 + t2; \
         d2 = t0 - t2; \
         d1 = t1 + t3; \
         d3 = t1 - t3; \
}

#define abs(x) ( ( (x) < 0 ) ? -(x) : (x) )
#define min(x,y) ( (x) < (y) ? (x) : (y) )

//////////////////
// CONSTANT MEMORY
//////////////////
__device__ __constant__ int ipa[TOTAL_MODES] = {0, 0, 32, 26, 21, 17, 13, 9, 5, 2, 0, -2, -5, -9, -13, -17, -21, -26, -32, -26, -21, -17, -13, -9, -5, -2, 0, 2, 5, 9, 13, 17, 21, 26, 32};
__device__ __constant__ int ia[IA_MODES] = {-4096, -1638, -910, -630, -482, -390, -315, -256, -315, -390, -482, -630, -910, -1638, -4096};

__device__ int sumArray(uint8_t *array, uint8_t start, uint8_t end)
{

    int result=0;
    
    for ( int counter = start; counter <= end; counter++ ){
        result += array[counter];
 

           //if(threadIdx.x == 0 && threadIdx.y==0 && blockIdx.x == 1 && blockIdx.y == 0)
           //printf("%d\n",result);
    }

    //if(threadIdx.x == 0 && threadIdx.y ==0 && blockIdx.x == 0 && blockIdx.y == 0)
      //  printf("returning result : %d\n",result);
     
  
    return result;

} // End of sumArray()

__device__ uint8_t clip3(uint8_t x, uint8_t y, uint8_t z)
{

    if ( z < x )
        return x;
    else if ( z > y )
        return y;
    else
        return z;

} // End of clip3()

__device__ uint8_t clip1Y(uint8_t x)
{

    uint8_t ret = clip3(0, ( 1 << BITDEPTHY ) - 1, x);

    return ret;

} // End of clip1Y()

__device__ sum2_t abs2(sum2_t a)
{
    sum2_t s = ((a >> (BITS_PER_SUM - 1)) & (((sum2_t)1 << BITS_PER_SUM) + 1)) * ((sum_t)-1);
    return (a + s) ^ s;
}

__device__ void sort(int32_t*  input_values)
{
        for(int i =0;i<TOTAL_MODES;i++)
        {
            int j=i;
            while(j>0 && (input_values[j] < input_values[j-1]))
            {
                int32_t temp=input_values[j];
                input_values[j]=input_values[j-1];
                input_values[j-1]=temp;
                j--;
            }
        }    
      
        if(threadIdx.x ==0 && threadIdx.y==0 && blockIdx.x == 0 && blockIdx.y == 0){
         //  printf("HERE IN THE SORT FUNCTION\n");
     //      for(int i=0;i<35;i++) printf("sorted values : %d\n",input_values[i]);
        }

            
} // End of sort()

__device__ void extract(int32_t *sorted_values, uint8_t *modes)
{

   for ( int counter = 0; counter < TOTAL_MODES; counter++)
   {
         uint8_t mode = (sorted_values[counter] & 0XFF);
         int32_t value = (sorted_values[counter] >> 8);

         sorted_values[counter] = value;
         modes[counter] = mode;

       //  if(blockIdx.x ==0 && blockIdx.y == 0)
         //    printf("res_counter : %d\n",res[counter]);
   }

   if(threadIdx.x ==0 && threadIdx.y==0 && blockIdx.x == 0 && blockIdx.y == 0){
         //  for(int i=0;i<35;i++) printf("%d\n",sorted_values[i]);
           
        }


} // End of extract()

//////////////////////////////////////////////////////
//////////////////////////////////////////////////////
//////////////  KERNEL FUNCTION  /////////////////////
//////////////////////////////////////////////////////
//////////////////////////////////////////////////////
/*
__global__ void hevcPredictionKernel(uint8_t *y, uint8_t *cr, uint8_t *cb, int32_t *res_y, int32_t *res_cr, int32_t *res_cb, uint8_t *y_modes, uint8_t *cr_modes, uint8_t *cb_modes, int height, int width)
{
 printf("\nYUP I AM HERE\n");

}
*/

__global__ void hevcPredictionKernel(uint8_t *y, uint8_t *cr, uint8_t *cb, int32_t *res_y, int32_t *res_cr, int32_t *res_cb, uint8_t *y_modes, uint8_t *cr_modes, uint8_t *cb_modes, int height, int width)
{

    // Thread indices, Block Indices and Dimensions
    uint8_t bsize = blockDim.x;

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Thread Index to Data Index Mapping
    int col = tx + blockDim.x * bx; 
    int row = ty + blockDim.y * by;

    if ( 0 == tx && 0 == ty  && row == 0 && col == 0)
        printf("\n YUP I AM HERE \n");

    // Shared neighbour memory
    int neighbourArraySize = (bsize * TWO) + ONE;

    int bitDepthY=BITDEPTHY;
    int bitDepthC=BITDEPTHC;

    int rowToBeLoaded=0;
    int colToBeLoaded=0;
    int var = 1;
    int var1 =  0;
    
    /////////
    // Neighbour Array
    ////////
    // y is vertical array that has the extra element that is [-1][-1]
    // x is horizontal component

    // Neigbour Array for luma component
    __device__ __shared__ uint8_t p_yy[MAX_BLOCK_SIZE*2+1];
    __device__ __shared__ uint8_t p_xy[MAX_BLOCK_SIZE*2+1];

    // Neighbour array for chroma component 
    __device__ __shared__ uint8_t p_ycr[MAX_BLOCK_SIZE*2+1];
    __device__ __shared__ uint8_t p_ycb[MAX_BLOCK_SIZE*2+1];
    __device__ __shared__ uint8_t p_xcr[MAX_BLOCK_SIZE*2+1];
    __device__ __shared__ uint8_t p_xcb[MAX_BLOCK_SIZE*2+1];
   
    // Pointer to neighbour elements in shared memory
    uint8_t *pyy = &p_yy[ONE];
    uint8_t *pxy = &p_xy[ZERO];
    uint8_t *pycr = &p_ycr[ONE];
    uint8_t *pxcr = &p_xcr[ZERO];
    uint8_t *pycb = &p_ycb[ONE];
    uint8_t *pxcb = &p_xcb[ZERO];

    // Points to the righ top most block for which all
    // the neighbour elements fall outside the image boundaries
    unsigned int fallOutside = 0;
    
    // This is to take care of the top right corner blocks in the grid
    // OPTIMIZATION
    if ( (0 == bx && 0 == by) )
         fallOutside = 1;

    /// DEBUG
    //if ( fallOutside )
        //printf("\nI AM FALLING OUTSIDE\n");

    /// DEBUG
    /*
    if ( blockIdx.x == 0  && by == 0 && tx == 0 && ty == 0 )
    {
    printf("\nINPUT MATRIX WIDTH: %d HEIGHT: %d\n", width, height);
    for ( int i = 0 ; i < width; i++)
    {
        for (int j = 0; j < height; j++ )
        {
             printf("\t%u", y[i*width+j]);
        }
        printf("\n");
    }
    }

    __syncthreads();
    */
    
    //////////////////////////////////
    //////////////////////////////////
    // Step 1: LOAD NEIGHBOUR ELEMENTS
    //////////////////////////////////
    //////////////////////////////////

    // Load into the shared memory from global memory
    // The loading is done based on a row basis

    // Load luma elements
    if ( ZERO == ty )
    {
        rowToBeLoaded=row-1;
        colToBeLoaded=col;

        /// DEBUG
        /*
        if ( var == bx && var1 == by )
           printf("\nRow: %d col: %d rowTO: %d colTO: %d\n", row, col, rowToBeLoaded, colToBeLoaded);
        */

        if((rowToBeLoaded>=0 && rowToBeLoaded<height && colToBeLoaded>=0 && colToBeLoaded<width) || fallOutside)
        { 
            pxy[tx] = (fallOutside == 1) ? (1 << (bitDepthY -1)) : y[(rowToBeLoaded*width)+colToBeLoaded];  
            pxcr[tx] = (fallOutside == 1) ? (1 << (bitDepthC - 1)) : cr[(rowToBeLoaded*width)+colToBeLoaded];
            pxcb[tx] = (fallOutside == 1) ? (1 << (bitDepthC - 1)) : cb[(rowToBeLoaded*width)+colToBeLoaded];
        }
    }
    else if ( ONE == ty )
    {
        rowToBeLoaded=row-2;
        colToBeLoaded=col+blockDim.x;

        /// DEBUG
        /*
        if ( var == bx && var1 == by )
           printf("\nRow: %d col: %d rowTO: %d colTO: %d\n", row, col, rowToBeLoaded, colToBeLoaded);
        */

        if((rowToBeLoaded>=0 && rowToBeLoaded<height && colToBeLoaded>=0 && colToBeLoaded<width) || fallOutside)
        { 
    	    pxy[tx + bsize] = (fallOutside == 1) ? (1 << (bitDepthY - 1)) : y[(rowToBeLoaded*width)+colToBeLoaded];
            pxcr[tx + bsize] = (fallOutside == 1) ? (1 << (bitDepthC - 1)) : (cr[(rowToBeLoaded*width)+colToBeLoaded]);
            pxcb[tx + bsize] = (fallOutside == 1) ? (1 << (bitDepthC - 1)) : (cb[(rowToBeLoaded*width)+colToBeLoaded]);
        }
    }
    else if ( TWO == ty )
    {
        rowToBeLoaded=(row-2)+tx;
        colToBeLoaded=blockDim.x*blockIdx.x-1;

        /// DEBUG
        /*
        if ( var == bx && var1 == by )
           printf("\nRow: %d col: %d rowTO: %d colTO: %d\n", row, col, rowToBeLoaded, colToBeLoaded);
        */

        if((rowToBeLoaded>=0 && rowToBeLoaded<height && colToBeLoaded>=0 && colToBeLoaded<width) || fallOutside)
        { 
            pyy[tx] = (fallOutside == 1) ? (1 << (bitDepthY - 1)) : y[rowToBeLoaded*width + colToBeLoaded];
            pycr[tx] = (fallOutside == 1) ? (1 << (bitDepthC - 1)) : (cr[rowToBeLoaded*width + colToBeLoaded]);
            pycb[tx] = (fallOutside == 1) ? (1 << (bitDepthC - 1)) : (cb[rowToBeLoaded*width + colToBeLoaded]);
        }
    }
    else if ( THREE == ty )
    {
        rowToBeLoaded=(row+1)+tx;
        colToBeLoaded=blockIdx.x*blockDim.x-1;

        /// DEBUG
        /*
        if ( var == bx && var1 == by )
           printf("\nRow: %d col: %d rowTO: %d colTO: %d\n", row, col, rowToBeLoaded, colToBeLoaded);
        */

        if((rowToBeLoaded>=0 && rowToBeLoaded<height && colToBeLoaded>=0 && colToBeLoaded<width) || fallOutside)
        { 
            pyy[tx + bsize] = (fallOutside == 1) ? (1 << (bitDepthY - 1)) : y[rowToBeLoaded*width + colToBeLoaded];
            pycr[tx + bsize] = (fallOutside == 1) ? (1 << (bitDepthC - 1)) : (cr[rowToBeLoaded*width + colToBeLoaded]);
            pycb[tx + bsize] = (fallOutside == 1) ? (1 << (bitDepthC - 1)) : (cb[rowToBeLoaded *width + colToBeLoaded]);
        }
    }
    else
    {
        // Nothing to do here
    }
    
    // This is to load the extra guy in the neighbour element array
    // who is not filled by the threads in the current block
    // i.e. the extra element in the pyy, pycr, pycb array
    if ( 0 == tx && 0 == ty ) 
    {
        if ( ! ((0 == bx) || (0 == by)) )
        {
            // this should have been pyy[MINUS]
            rowToBeLoaded=row-1;
            colToBeLoaded=col-1;
            
            if(rowToBeLoaded>=0 && rowToBeLoaded<height && colToBeLoaded>=0 && colToBeLoaded<width)
            { 
                pyy[MINUS] = y[(rowToBeLoaded-1)*width + (colToBeLoaded-1)];
                pycr[MINUS] = y[(rowToBeLoaded-1)*width + (colToBeLoaded-1)];
                pycb[MINUS] = y[(rowToBeLoaded-1)*width + (colToBeLoaded-1)];
            }
        } // End of if ( ! ((0 == bx) || (0 == by)) )
        if ( fallOutside) 
        {
            pyy[MINUS] = 1 << (bitDepthY - 1);
            pycr[MINUS] = 1 << (bitDepthC - 1);
            pycb[MINUS] = 1 << (bitDepthC - 1);
        }
    } // End of if ( 0 == tx && 0 == ty )

    __syncthreads();

    /// DEBUG
    /*
    if ( blockIdx.x == var && blockIdx.y == var1 && tx == 0 && ty == 0 )
    {
    printf("\nPREDICTED MATRIX - PYY\n");
    for ( int i = 0 ; i < 2*bsize+1; i++)
    {
             printf("\t%u", p_yy[i]);
    }
    printf("\nPREDICTED MATRIX - PXY\n");
    for ( int i = 0 ; i < 2*bsize; i++)
    {
             printf("\t%u", p_xy[i]);
    }
    }
    */
 

 
    //////////////////////////
    //////////////////////////
    // Step 2: First Filtering
    //////////////////////////
    //////////////////////////
    
    if ( ZERO == tx && ZERO == ty )
    {

        if (by==(gridDim.y-1))
        {
            if(bx==ZERO)
            {
                for(int i=0;i<neighbourArraySize-1;i++)
                {
                    pyy[i]=pxy[ZERO];
                    pycr[i] = pxcr[ZERO];
                    pycb[i] = pxcb[ZERO];
                }
                pyy[MINUS] = pxy[ZERO];
                pycr[MINUS] = pxcr[ZERO];
                pycb[MINUS] = pxcb[ZERO];
            }
            else
            {
                for(int i=bsize;i<(2*bsize);i++)
                {
                    pyy[i]=pyy[bsize-ONE];
                    pycr[i] = pycr[bsize-ONE];
                    pycb[i] = pycb[bsize-ONE];
                }
            }
         } // End of if (by==(gridDim.y-1))
         if(0==by && !fallOutside)
         {
             pyy[MINUS]=pyy[ZERO];
             pycr[MINUS] = pycr[ZERO];
             pycb[MINUS] = pycb[ZERO];
             for(int i=0;i<2*bsize;i++)
             {
                 pxy[i]=pyy[MINUS];
                 pxcr[i]=pycr[MINUS];
                 pxcb[i]=pycb[MINUS];
             }
         } // End of if ( 0 == by )
         if((bx == (gridDim.x - 1)) && (0 != by))
         {
             for ( int i = bsize; i < (2 * bsize); i++ )
             {
                 pxy[i] = pxy[bsize - 1];
                 pxcr[i] = pxcr[bsize - 1];
                 pxcb[i] = pxcb[bsize - 1];
             }
         }
    } // End of if ( ZERO == tx && ZERO == ty )
  
    __syncthreads();

    /// DEBUG
    /* 
    if ( blockIdx.x == var && blockIdx.y == var1 && tx == 0 && ty == 0 )
    {
    printf("\nPREDICTED MATRIX - PYY\n");
    for ( int i = 0 ; i < 2*bsize+1; i++)
    {
             printf("\t%u", p_yy[i]);
    }
    printf("\nPREDICTED MATRIX - PXY\n");
    for ( int i = 0 ; i < 2*bsize; i++)
    {
             printf("\t%u", p_xy[i]);
    }
    }
    */
    


    /////////////////////////////////////////////////
    /////////////////////////////////////////////////
    // STEP 3 : MODE COMPUTATION AND SECOND FILTERING
    /////////////////////////////////////////////////
    /////////////////////////////////////////////////

    // TO DO
    /////////
    // Second Filtered neighbour array
    /////////
    __device__ __shared__ uint8_t pf_yy[MAX_BLOCK_SIZE*2+1];
    __device__ __shared__ uint8_t pf_xy[MAX_BLOCK_SIZE*2+1];   
  
    ////////
    // Predicted pixels
    ///////
    __device__ __shared__ uint8_t predSamplesY[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];
    __device__ __shared__ uint8_t predSamplesCr[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];
    __device__ __shared__ uint8_t predSamplesCb[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];

    // Pointer to predicted pixels
    uint8_t *pfyy = &pf_yy[ONE];
    uint8_t *pfxy = &pf_xy[ZERO];

    //////
    // Hadamard shared memory
    //////
    __device__ __shared__ sum2_t ay[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];
    __device__ __shared__ sum2_t acr[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];
    __device__ __shared__ sum2_t acb[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];

    __device__ __shared__ sum2_t hby[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE/2];
    __device__ __shared__ sum2_t bcr[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE/2];
    __device__ __shared__ sum2_t bcb[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE/2];
    __device__ __shared__ sum2_t had8y[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE/2];
    __device__ __shared__ sum2_t had8cr[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE/2];
    __device__ __shared__ sum2_t had8cb[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE/2];

    __device__ __shared__ int32_t y_satd_shared[TOTAL_MODES];
    __device__ __shared__ int32_t cr_satd_shared[TOTAL_MODES];
    __device__ __shared__ int32_t cb_satd_shared[TOTAL_MODES];
    __device__ __shared__ uint8_t y_modes_shared[TOTAL_MODES];
    __device__ __shared__ uint8_t cr_modes_shared[TOTAL_MODES];
    __device__ __shared__ uint8_t cb_modes_shared[TOTAL_MODES];
   
    
 
    // Loop through all modes
    for(int mode =0;mode <35;mode++)
    {
        // if the  computed value of  filterFlag==1, use the filtered array pF instead of p for intra prediction.
        int filterFlag=0;
        int biIntFlag= 0;
   
        if(ty==0 && tx==0)
        { 
            //////////////
            // FILTER FLAG
            //////////////
            if(mode==DC_MODE || bsize==4)
            {
                filterFlag=0;
            }
            else
            {
                int minDistVerHor = min(abs(mode-26),abs(mode-10));
                int intraHorVerDistThres;
   			 
                if(bsize==8)
                {
                    intraHorVerDistThres=7;
                }
                else if(bsize==16)
                {
       		    intraHorVerDistThres=1;
                }
  		else if(bsize==32)
                {
       	            intraHorVerDistThres=0;
                }
                else
                {
                    // Nothing to do`
                }
		if(minDistVerHor>intraHorVerDistThres)
                {
        	    filterFlag=1;
                }
                else
                {
                    filterFlag = 0;
                }
            } // End of else of if ( mode == DC_MODE || bsize == 4 )
           
            if(filterFlag==1)
            {
                /////////////
                // B INT FLAG
                /////////////
                if(bsize==32 && ( abs ( pyy[-1] + pxy[bsize*2-1] - (2*pxy[bsize-1]) ) < (1<<(bitDepthY-5) ) ) && ( abs ( pyy[-1] + pyy[bsize*2-1] - (2*pyy[bsize-1]) ) < (1<<(bitDepthY-5) ) ))
                {
                    biIntFlag=1;
                }
                else
                {
                    biIntFlag = 0;
                }
            } // End of if ( 1 == filterFlag )

     
            ///////////////////
            // SECOND FILTERING
            ///////////////////
            if(biIntFlag==1)
            {
                pfyy[MINUS]=pyy[MINUS];
                for(int i=0;i<(bsize*2-1);i++)
                {
                    pfyy[i]=((63-i)*pyy[MINUS]+(i+1)*pyy[63]+32)>>6; 
                }
                pfyy[63]=pyy[63];
                for(int i=0;i<(bsize*2-1);i++)
                {
                    pfxy[i]=((63-i)*pyy[MINUS]+(i+1)*pxy[63]+32)>>6;
                }
                pfxy[63]=pxy[63];
            } // End of if ( 1 == biIntFlag )
            else
            {
                pfyy[MINUS]=(pyy[ZERO]+2*pyy[MINUS]+pxy[ZERO]+2)>>2;
                for(int i=0;i<(bsize*2-1);i++)
                {
                    pfyy[i]=(pyy[i+1]+2*pyy[i]+pyy[i-1]+2)>>2;
                }
                pfyy[bsize*2-1]=pyy[bsize*2-1];
                pfxy[0] = (pyy[MINUS] + 2 * pxy[ZERO] + pxy[ONE] + 2) >> 2;
                for(int i=1;i<(bsize*2-1);i++)
                {
                    pfxy[i]=(pxy[i-1]+2*pxy[i]+pxy[i+1]+2)>>2;
                }
                pfxy[bsize*2-1]=pxy[bsize*2-1];
           } // End of else of if ( 1 -- biIntFlag )

       } // End of if(ty==0 && tx==0)
    
        __syncthreads();

        //////////////
        // Switch pointer to pfyy or p_yy
        // Switch pointer to pfxy or p_xy
        /////////////
        uint8_t *selyy, *selxy; 
        if(filterFlag==1)
        {
            if(bx ==var && by==var1 && tx==0 && ty==0 && mode == PLANAR_MODE)
                  printf("selected pf as neighbor");
            selyy=&pf_yy[ONE];
            selxy=&pf_xy[ZERO];
        }
        else
        {

            if(bx ==var && by==var1 && tx==0 && ty==0 && mode == PLANAR_MODE)
                  printf("selected p as neighbor");
            selyy=pyy;
            selxy=pxy;
        }   

        __device__ __shared__ uint8_t ref_Y[3*MAX_BLOCK_SIZE+1];
        __device__ __shared__ uint8_t ref_Cr[3*MAX_BLOCK_SIZE+1];
        __device__ __shared__ uint8_t ref_Cb[3*MAX_BLOCK_SIZE+1];

        // Pointer to ref arrays
        uint8_t *refY = &ref_Y[4];
        uint8_t *refCr = &ref_Cr[4];
        uint8_t *refCb = &ref_Cb[4];

        // OPTIMIZATION making iIdx and IFact as matrices
        __device__ __shared__ int iIdx[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];
        __device__ __shared__ int iFact[MAX_BLOCK_SIZE][MAX_BLOCK_SIZE];


        ////////////////////
        // MODE: PLANAR MODE
        ////////////////////
        // TO DO : is this ty tx
        if(mode==PLANAR_MODE)
        {

            float logValue = log2f(bsize);
            int intLog = (int) logValue+1;
             
            if(bx==var && by==var1 && tx==0 && ty==0){
                    printf("\nlog value is %d\n",intLog);
                    printf("selected pointer vaalue is selyy[ty]  %d\n",selyy[ty]);
                    printf("selected pointer vaalue is selxy[bsize]%d\n",selxy[bsize]);
                    printf("selected pointer vaalue is selxy[tx]%d\n",selxy[tx]);
                    printf("selected pointer vaalue is selyy[bsize]%d\n",selyy[bsize]);
            }
            predSamplesY[ty][tx]=((bsize-1-tx)*selyy[ty]+(tx+1)*selxy[bsize]+(bsize-1-ty)*selxy[tx]+(ty+1)*selyy[bsize]+bsize)>>intLog; //TO_DO: Replace logarithmic with appropriate C function    
            predSamplesCr[ty][tx]=((bsize-1-tx)*pycr[ty]+(tx+1)*pxcr[bsize]+(bsize-1-ty)*pxcr[tx]+(ty+1)*pycr[bsize]+bsize)>>intLog; //TO_DO: Replace logarithmic with appropriate C function    
            predSamplesCb[ty][tx]=((bsize-1-tx)*pycb[ty]+(tx+1)*pxcb[bsize]+(bsize-1-ty)*pxcb[tx]+(ty+1)*pycb[bsize]+bsize)>>intLog; //TO_DO: Replace logarithmic with appropriate C function    


       
    
        }

        ////////////////
        // MODE: DC MODE
        ////////////////
        else if ( DC_MODE == mode )
        {

            __shared__ uint8_t dcValY;
            __shared__ uint8_t dcValCr;
            __shared__ uint8_t dcValCb;
 
            __shared__ int firstSumY ;
            __shared__ int secondSumY ;
            __shared__ int firstSumCr ;
            __shared__ int secondSumCr ;
            __shared__ int firstSumCb ;
            __shared__ int secondSumCb ;

            //OPTIMIZATION 
            if ( 0 == tx && 0 == ty )
            {
                firstSumY = sumArray(selxy, 0, bsize - 1);
                if ( bx == var && by == var1 )
                    printf("\n FIRST SUMM Y PRINTING %d\n", firstSumY);
            }
            else if ( 1 == tx && 0 == ty )
            {
                secondSumY = sumArray(selyy, 0, bsize - 1);
            }
            else if ( 2 == tx && 0 == ty ) 
            {
                firstSumCr = sumArray(pxcr, 0, bsize - 1);
            } 
            else if ( 3 == tx && 0 == ty )
            {
                secondSumCr = sumArray(pycr, 0, bsize - 1);
                if(bsize==4)
                {
                    firstSumCb = sumArray(pxcb, 0, bsize - 1);
                    secondSumCb = sumArray(pycb, 0, bsize - 1);
                }     
            }
            else if ( 4 == tx && 0 == ty && bsize!=4 )
            {
                firstSumCb = sumArray(pxcb, 0, bsize - 1);
            } 
            else if ( 5 == tx && 0 == ty && bsize!=4)
            {
                secondSumCb = sumArray(pycb, 0, bsize - 1);
            }

            __syncthreads(); 
 
            if ( 0 == tx && 0 == ty )
            {
                //DEBUG
                /*
                if(bx==var && by == var1){
                       printf("firstsumY : %d , secondsumy : %d, log value %d\n",firstSumY,secondSumY,((int)log2f((float)bsize)+1));
                }
                */

                dcValY = (firstSumY + secondSumY + bsize) >> ((int)log2f((float)bsize)+1);
                //DEBUG
                /*
                if(bx==var && by == var1){
                        printf("first dcvaly : %d\n",dcValY);
                }
                */
            }

            else if ( 1 == tx && 0 == ty )
            {
                dcValCr = (firstSumCr + secondSumCr + bsize) >> ((int)log2f((float)bsize)+1);
            }

            else if ( 2 == tx && 0 == ty )
            {
                dcValCb = (firstSumCb + secondSumCb + bsize) >> ((int)log2f((float)bsize)+1);
            }

            __syncthreads();

            if ( bsize < 32 )
            {
                //Apply following changes to predSamples only for luma channel
                if(ZERO == ty && ZERO == tx){
                    predSamplesY[0][0]=(selyy[ZERO]+2*dcValY+selxy[0]+2)>>2;
                    if(bx==var && by == var1){
                          printf("dcvaly :%d, predsamplesy : %d\n",dcValY,predSamplesY[0][0]);
                    }
                }
          
                if(ZERO == ty && tx>=ONE && tx<=bsize-1){ 
                    if(tx==1 && bx==var && by==var1)
                          printf("Loading selxy[tx] as %u and dcValY as %d",selxy[tx],dcValY);
            	    predSamplesY[ty][tx]=(selxy[tx]+3*dcValY+2)>>2;
                }
                if(ZERO == tx && ty>=ONE && ty<=bsize-1) 
            	    predSamplesY[ty][tx]=(selyy[ty]+3*dcValY+2)>>2;
                
                 if(tx >0 && ty >0)
                      predSamplesY[ty][tx]=dcValY;
            } // End of if ( bsize < 32 )
            else 
            {
                //For cr and cb, set dcValue as all value for predSamples of cr and cb  
                      predSamplesY[ty][tx] = dcValY;
                      predSamplesCr[ty][tx]=dcValCr;
                      predSamplesCb[ty][tx]=dcValCb;
            } // End of else of if ( bsize < 32 )

        } // End of else if ( DC_MODE == mode )

        ///////////////
        // ANGULAR MODE
        ///////////////

        else if ( mode >= ANGULAR_18 )
        {

            // OPTIMIZATION 
            if ( bsize == 4 )
            {
                if ( 0 == ty ) 
                {
                    if(tx==0){
                        refY[0]=selyy[MINUS];
                        refCr[0]=pycr[MINUS];
                        refCb[0]=pycb[MINUS];
                    }
                    else{
                        refY[tx] = selxy[-1 + tx];
                        refCr[tx] = pxcr[-1 + tx]; 
                        refCb[tx] = pxcb[-1 + tx];
                    }
                    if ( 0 == tx )
                    {
                        refY[bsize+tx] = selxy[-1 + (tx + bsize)];
                        refCr[bsize+tx] = pxcr[-1 + (tx + bsize)];
                        refCb[bsize+tx] = pxcb[-1 + (tx + bsize)];
                    }
                }

                if (ipa[mode] < 0) 
                {
                    if ( ((bsize * ipa[mode]) >> 5) < -1 )
                    {
                        if ( 0 == tx && 0 == ty )
                        {
                            int range = ( bsize * ipa[mode] ) >> 5;
                            for ( int counter = -1; counter >= range; counter-- )
                            {
                                refY[counter] = selyy[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                                refCr[counter] = pycr[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                                refCb[counter] = pycb[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                            }
                        }
                        /*if ( 1 == ty )
                        {
                            refY[-(tx + 1)] = selyy[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                        }
                        if ( 2 == ty ) 
                        {
                            refCr[-(tx + 1)] = pycr[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                            refCb[-(tx + 1)] = pycb[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                        } */
                    } // End of if ( ((bsize * ipa[mode]) >> 5) < -1 )
                } // End of if (ipa[mode] < 0)
                else 
                {
                    if ( 3 == ty )
                    {
                        refY[tx + bsize + 1] = selxy[-1 + tx + bsize + 1];
                        refCr[tx + bsize + 1] = pxcr[-1 + tx + bsize + 1];
                        refCb[tx + bsize + 1] = pxcb[-1 + tx + bsize + 1];
                    }
                } // End of else of if (ipa[mode] < 0)

            } // End of if ( bsize == 4 )
            else 
            {
                if ( 0 == ty ) 
                {
                    if(tx==0){
                        refY[0]=selyy[MINUS];
                    }
                    else
                        refY[tx] = selxy[-1 + tx];
                    if ( 0 == tx )
                        refY[bsize + tx] = selxy[-1 + (tx + bsize)];
                }
                if ( 1 == ty ) 
                {
                    if(tx==0){
                        refCr[0]=pycr[MINUS];
                    }
                    else
                       refCr[tx] = pxcr[-1 + tx];
                    if ( 0 == tx )
                        refCr[bsize+tx] = pxcr[-1 + (tx + bsize)];
                } 
                if ( 2 == ty ) 
                {
                    if(tx==0){
                        refCb[0]=pycb[MINUS];
                    }
                    else
                        refCb[tx] = pxcb[-1 + tx];
                    if ( 0 == tx )
                        refCb[bsize+tx] = pxcb[-1 + (tx + bsize)];
                } 
                if (ipa[mode] < 0)
                {
                    if ( ((bsize * ipa[mode]) >> 5) < -1 )
                    {
                        if ( 0 == tx && 0 == ty )
                        {
                            int range = ( bsize * ipa[mode] ) >> 5;
                            for ( int counter = -1; counter >= range; counter-- )
                            {
                                refY[counter] = selyy[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                                refCr[counter] = pycr[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                                refCb[counter] = pycb[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                            }
                        }
                        /*if ( 3 == ty )
                        {
                            refY[-(tx + 1)] = selyy[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                        }
                        if ( 4 == ty )
                        {
                            refCr[-(tx + 1)] = pycr[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                        }
                        if ( 5 == ty )
                        {
                            refCb[-(tx + 1)] = pycb[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                        }*/
                    } // End of if ( ((bsize * ipa[mode]) >> 5) < -1 )
                } // End of if (ipa[mode] < 0)
                else
                {
                    if ( 6 == ty )
                        refY[tx + bsize + 1] = selxy[-1 + tx + bsize + 1];
                    if ( 7 == ty )
                    {
                        refCr[tx + bsize + 1] = pxcr[-1 + tx + bsize + 1];
                        if ( bsize == 8 )
                            refCb[tx + bsize + 1] = pxcb[-1 + tx + bsize + 1];
                    }
                    if ( 8 == ty && bsize != 8 )
                    {
                        refCb[tx + bsize + 1] = pxcb[-1 + tx + bsize + 1];
                    }
                } // End of else of if (ipa[mode] < 0)
                
            } // End of else of if ( bsize == 4 )

            // Load iIdx and iFact
            iIdx[ty][tx] = ((ty+1) * ipa[mode]) >> 5;
            iFact[ty][tx] = ((ty+1) * ipa[mode]) & 31;

            if ( iFact[ty][tx] != 0 )
            {
                predSamplesY[ty][tx] = ((32 - iFact[ty][tx]) * refY[tx + iIdx[ty][tx] + 1] + iFact[ty][tx] * refY[tx + iIdx[ty][tx] + 2] + 16) >> 5;
                predSamplesCr[ty][tx] = ((32 - iFact[ty][tx]) * refCr[tx + iIdx[ty][tx] + 1] + iFact[ty][tx] * refCr[tx + iIdx[ty][tx] + 2] + 16) >> 5;
                predSamplesCb[ty][tx] = ((32 - iFact[ty][tx]) * refCb[tx + iIdx[ty][tx] + 1] + iFact[ty][tx] * refCb[tx + iIdx[ty][tx] + 2] + 16) >> 5;
            } 
            else
            {
                predSamplesY[ty][tx] = refY[tx + iIdx[ty][tx] + 1];
                predSamplesCr[ty][tx] = refCr[tx + iIdx[ty][tx] + 1];
                predSamplesCb[ty][tx] = refCb[tx + iIdx[ty][tx] + 1];
            }

            if ( mode == ANGULAR_26 && bsize < 32 )
            {
                if ( 0 == tx ) 
                {
                    uint8_t param = selxy[tx] + ((selyy[ty] - selyy[MINUS]) >> 1);
                    predSamplesY[ty][tx] = clip1Y(param);
                }


            } // End of if ( mode == ANGULAR_26 && bsize < 32 )

        } // End of else if ( mode >= ANGULAR_18 )

        else if ( mode > DC_MODE && mode < ANGULAR_18 )
        {
            if ( 4 == bsize )
            {

                if ( 0 == ty )
                {
                    refY[tx] = selyy[-1 + tx];
                    refCr[tx] = pycr[-1 + tx];
                    refCb[tx] = pycb[-1 + tx];
                    if ( 0 == tx )
                    {
                        refY[bsize+tx] = selyy[-1 + (tx + bsize)];
                        refCr[bsize+tx] = pycr[-1 + (tx + bsize)];
                        refCb[bsize+tx] = pycb[-1 + (tx + bsize)];
                    }
                }

                if (ipa[mode] < 0)
                {
                    if ( ((bsize * ipa[mode]) >> 5) < -1 )
                    {
                        if ( 0 == tx && 0 == ty )
                        {
                            int range = ( bsize * ipa[mode] ) >> 5;
                            for ( int counter = -1; counter >= range; counter-- )
                            {
                                refY[counter] = selyy[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                                refCr[counter] = pycr[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                                refCb[counter] = pycb[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                            }
                        }
                        /*if ( 1 == ty )
                        {
                            if((-1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8)<0)
                                  refY[-(tx + 1)] = selyy[MINUS];
                            else
                               refY[-(tx + 1)] = selxy[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                        }
                        if ( 2 == ty )
                        {
                            if((-1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8)<0){
                                  refCr[-(tx + 1)] = pycr[MINUS];
                                  refCb[-(tx + 1)] = pycb[MINUS];
                            }
                            else{
                                 refCr[-(tx + 1)] = pxcr[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                                 refCb[-(tx + 1)] = pxcb[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                            }
                        }*/
                    } // End of if ( ((bsize * ipa[mode]) >> 5) < -1 )
                } // End of if (ipa[mode] < 0)
                else
                {
                    if ( 3 == ty )
                    {
                        refY[tx + bsize + 1] = selyy[-1 + tx + bsize + 1];
                        refCr[tx + bsize + 1] = pycr[-1 + tx + bsize + 1];
                        refCb[tx + bsize + 1] = pycb[-1 + tx + bsize + 1];
                    }
                } // End of else of if (ipa[mode] < 0)

            } // End of if ( 4 == bsize )
            else
            {
 
                if ( 0 == ty )
                {
                    refY[tx] = selyy[-1 + tx];
                    if ( 0 == tx )
                        refY[bsize + tx] = selyy[-1 + (tx + bsize)];
                }
                if ( 1 == ty )
                {
                    refCr[tx] = pycr[-1 + tx];
                    if ( 0 == tx )
                        refCr[bsize+tx] = pycr[-1 + (tx + bsize)];
                }
                if ( 2 == ty )
                {
                    refCb[tx] = pycb[-1 + tx];
                    if ( 0 == tx )
                        refCb[bsize+tx] = pycb[-1 + (tx + bsize)];
                }
                if (ipa[mode] < 0)
                {
                    if ( ((bsize * ipa[mode]) >> 5) < -1 )
                    {
                        if ( 0 == tx && 0 == ty )
                        {
                            int range = ( bsize * ipa[mode] ) >> 5;
                            for ( int counter = -1; counter >= range; counter-- )
                            {
                                refY[counter] = selyy[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                                refCr[counter] = pycr[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                                refCb[counter] = pycb[ -1 + (( counter * ia[mode-11] + 128) >> 8) ];
                            }
                        }
                        /*
                        if ( 3 == ty )
                        {
                            if((-1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8)<0)
                                refY[-(tx + 1)] = selyy[MINUS];
                            else
                                refY[-(tx + 1)] = selxy[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                        }
                        if ( 4 == ty )
                        {
                            if((-1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8)<0)
                                refCr[-(tx + 1)] = pycr[MINUS];
                            else
                                refCr[-(tx + 1)] = pxcr[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                        }
                        if ( 5 == ty )
                        {
                            if((-1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8)<0)
                                refCb[-(tx + 1)] = pycb[MINUS];
                            else
                                refCb[-(tx + 1)] = pxcb[ -1 + (( tx + 1 ) * ia[mode-11] + 128) >> 8];
                        }
                        */
                    } // End of if ( ((bsize * ipa[mode]) >> 5) < -1 )
                } // End of if (ipa[mode] < 0)
                else
                {
                    if ( 6 == ty )
                        refY[tx + bsize + 1] = selyy[-1 + tx + bsize + 1];
                    if ( 7 == ty )
                    {
                        refCr[tx + bsize + 1] = pycr[-1 + tx + bsize + 1];
                        if ( bsize == 8 )
                            refCb[tx + bsize + 1] = pycb[-1 + tx + bsize + 1];
                    }
                    if ( 8 == ty && bsize != 8 )
                    {
                        refCb[tx + bsize + 1] = pycb[-1 + tx + bsize + 1];
                    }
                } // End of else of if (ipa[mode] < 0)
                
            } // End of else of if ( 4 == bsize )

            // Load iIdx and iFact
            iIdx[ty][tx] = ( (tx + 1) * ipa[mode] ) >> 5;
            iFact[ty][tx] = ( (tx + 1) * ipa[mode] ) & 31;

            if ( iFact[ty][tx] != 0 )
            {
                predSamplesY[ty][tx] = ((32 - iFact[ty][tx]) * refY[ty + iIdx[ty][tx] + 1] + iFact[ty][tx] * refY[ty + iIdx[ty][tx] + 2] + 16) >> 5;
                predSamplesCr[ty][tx] = ((32 - iFact[ty][tx]) * refCr[ty + iIdx[ty][tx] + 1] + iFact[ty][tx] * refCr[ty + iIdx[ty][tx] + 2] + 16) >> 5;
                predSamplesCb[ty][tx] = ((32 - iFact[ty][tx]) * refCb[ty + iIdx[ty][tx] + 1] + iFact[ty][tx] * refCb[ty + iIdx[ty][tx] + 2] + 16) >> 5;
            }
            else
            {
                predSamplesY[ty][tx] = refY[ty + iIdx[ty][tx] + 1];
                predSamplesCr[ty][tx] = refCr[ty + iIdx[ty][tx] + 1];
                predSamplesCb[ty][tx] = refCb[ty + iIdx[ty][tx] + 1];
            }

            if ( mode == ANGULAR_10 && bsize < 32 )
            {
                if ( 0 == tx )
                    predSamplesY[ty][tx] = clip1Y(( (selyy[ty]) + ((selxy[tx]-selyy[MINUS])>>1) ));


            } // End of if ( mode == ANGULAR_10 && bsize < 32 )

        } // End of else if ( mode > ANGULAR_1 && mode < ANGULAR_18 )

        __syncthreads(); 

        if(bx==var && by==var1 && mode ==2 && tx==0 && ty==0){
               printf("For angular mode2 predSamplesY[0][0]:%d",predSamplesY[0][0]);
 
        } 
        ///////////////////
        // STEP 4: HADAMARD
        ///////////////////
        // finally calculation of SATD values for different modes
        // have A matrix which is a shared memory
        // all the threads fill the 'A' array
        if(bx==1 && by==0 && tx==0 && ty==0 && (mode == PLANAR_MODE || mode == DC_MODE || mode==2 || mode==3 || mode==4 || mode==5 || mode==6 ) )
        {  
            for(int i=0;i<bsize;i++)
            {  
               for(int j=0;j<bsize;j++){
                  //   printf("before HADAMARD for mode %d : predicted pixel %d\n",mode,predSamplesY[i][j]);
                }
            }
         }      

//////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////// SATD 4 ///////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////


        if(bsize == 4)
        {
               
           // everybody computes the difference of pixels
         
           ay[ty][tx]  = y[row*width + col] - predSamplesY[ty][tx];
           acr[ty][tx] = abs2(predSamplesCr[ty][tx] - cr[row*width + col]);
           acb[ty][tx] = abs2(predSamplesCb[ty][tx] - cb[row*width + col]);
          
           __syncthreads();
           
           // construct the B-matrix : 8 threads are working
           if(tx < 2)
           {
               hby[ty][tx] = ay[ty][2*tx] + ay[ty][2*tx + 1] + ((ay[ty][2*tx] - ay[ty][2*tx + 1]) << BITS_PER_SUM);
               bcr[ty][tx] = acr[ty][2*tx] + acr[ty][2*tx + 1] +((acr[ty][2*tx] - acr[ty][2*tx+1]) << BITS_PER_SUM);
               bcb[ty][tx] = acb[ty][2*tx] + acb[ty][2*tx + 1] + ((acb[ty][2*tx] - acb[ty][2*tx+1]) << BITS_PER_SUM);
           }

           __syncthreads();

           if(tx == 0)
           {
               sum2_t sumColumn=hby[ty][tx]+hby[ty][tx+1];
               sum2_t diffColumn=hby[ty][tx]-hby[ty][tx+1];
               hby[ty][0]=sumColumn;
               hby[ty][1]=diffColumn;
           }
           __syncthreads();

           if(tx == 3)
           {  
               // 4 threads work to calculate the value
              if(ty == 0)
              {

                  sum2_t a0; // = ay[3][0];
                  sum2_t a1; // = ay[3][1];
                  sum2_t a2; // = ay[3][2];
                  sum2_t a3; // = ay[3][3];

                  sum2_t sumy  = 0 ;
                  
                
    
                  for (int i = 0; i < 2; i++)
                  {
                      HADAMARD4(a0,a1,a2,a3, hby[0][i], hby[1][i], hby[2][i], hby[3][i]);
                      a0 = abs2(a0) + abs2(a1) + abs2(a2) + abs2(a3);
                      sumy += ((sum_t)a0) + (a0 >> BITS_PER_SUM);
                  }
                     
                  sum2_t valued = sumy >> 1;
                  int intSum=(int)valued;
                  if(bx==0 && by==0 && ty==0 && (mode == 0 || mode==2 || mode==3 || mode ==4 || mode==5 || mode==6)  )
                        printf("computed value for mode %u is %d\n",mode,intSum);

                  sumy =  (((intSum) << 8) | mode) ;
                  y_satd_shared[mode] = sumy;      

              }
              if(ty == 1)
              {
                  sum2_t a0; // = ay[3][0];
                  sum2_t a1; // = ay[3][1];
                  sum2_t a2; // = ay[3][2];
                  sum2_t a3; // = ay[3][3];

                 
                  sum2_t sumcr = 0 ;
        

                  for (int i = 0; i < 2; i++)
                  {
                      HADAMARD4(a0,a1,a2,a3, bcr[0][i], bcr[1][i], bcr[2][i], bcr[3][i]);
                      a0 = abs2(a0) + abs2(a1) + abs2(a2) + abs2(a3);
                      sumcr += ((sum_t)a0) + (a0 >> BITS_PER_SUM);
                  }

                  sum2_t valued = sumcr >> 1;
                  int intSum=(int)valued;
                  if(bx==1 && by==0 && ty==0 && (mode == 0 || mode==2 || mode==3 || mode ==4 || mode==5 || mode==6)  )
                        printf("computed value for mode %u is %d\n",mode,intSum);
                  sumcr =  (((intSum) << 8) | mode) ;
                  cr_satd_shared[mode] = sumcr;
                  
                  
              }
              if(ty == 2)
              {
                  sum2_t a0; // = ay[3][0];
                  sum2_t a1; // = ay[3][1];
                  sum2_t a2; // = ay[3][2];
                  sum2_t a3; // = ay[3][3];


                  sum2_t sumcb = 0 ;


                  for (int i = 0; i < 2; i++)
                  {
                      HADAMARD4(a0,a1,a2,a3, bcb[0][i], bcb[1][i], bcb[2][i], bcb[3][i]);
                      a0 = abs2(a0) + abs2(a1) + abs2(a2) + abs2(a3);
                      sumcb += ((sum_t)a0) + (a0 >> BITS_PER_SUM);
                  }

                  sum2_t valued = sumcb >> 1;
                  int intSum=(int)valued;
                  if(bx==0 && by==0 && ty==0 && (mode == 0 || mode==2 || mode==3 || mode ==4 || mode==5 || mode==6)  ) 
                        printf("computed value for mode %u is %d\n",mode,intSum);
                  sumcb =  (((intSum) << 8) | mode) ;
                  cb_satd_shared[mode] = sumcb;
                  
              }
          }      
          // TO DO : Also store the sum values appropriately into the resultant array
          // TO DO : Write the same HADAMARD4 macro from the serial code

       }  // if ( 4 == bsize) // end of SATD 4 COMPUTATION
   
        
      // End of for(int mode =0;mode <35;mode++)
///////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////SATD 4 ENDS//////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////SATD 8////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////

      if(bsize == 8){
             // all 64 threads compute the difference     
             ay[ty][tx]  = y[row*width + col] - predSamplesY[ty][tx];
             acr[ty][tx] = cr[row*width + col] - predSamplesCr[ty][tx];
             acb[ty][tx] = cb[row*width + col] - predSamplesCb[ty][tx];
                  
             __syncthreads(); 
                
             // 32 threads work for the computation
             if(tx<4){
                  hby[ty][tx] = (ay[ty][2*tx] + ay[ty][2*tx+1]) + ((ay[ty][2*tx] - ay[ty][2*tx+1]) << BITS_PER_SUM);
                  bcr[ty][tx] = (acr[ty][2*tx] + acr[ty][2*tx + 1]) + ((acr[ty][2*tx] - acr[ty][2*tx+1]) << BITS_PER_SUM);
                  bcb[ty][tx] = (acb[ty][2*tx] + acb[ty][2*tx + 1]) + ((acb[ty][2*tx] - acb[ty][2*tx+1]) << BITS_PER_SUM);
             } 
                   
             __syncthreads();

             if(tx == 4 ){
                  HADAMARD4(had8y[ty][0], had8y[ty][1], had8y[ty][2], had8y[ty][3], hby[ty][0],  hby[ty][1], hby[ty][2], hby[ty][3]);
                  HADAMARD4(had8cr[ty][0], had8cr[ty][1], had8cr[ty][2], had8cr[ty][3], bcr[ty][0], bcr[ty][1], bcr[ty][2], bcr[ty][3]);
                  HADAMARD4(had8cb[ty][0], had8cb[ty][1], had8cb[ty][2], had8cb[ty][3], bcb[ty][0], bcb[ty][1], bcb[ty][2], bcb[ty][3]);
             }
            
             __syncthreads();

             if(tx == 5){
                   
                  if(ty==0){
                        sum2_t a0;
                        sum2_t a1;
                        sum2_t a2;
                        sum2_t a3;
                        sum2_t a4;
                        sum2_t a5;
                        sum2_t a6;
                        sum2_t a7;
                        sum2_t b0;
                        sum2_t sum = 0;
                        for (int i = 0; i < 4; i++)
                        {
                          HADAMARD4(a0, a1, a2, a3, had8y[0][i], had8y[1][i], had8y[2][i], had8y[3][i]);
                          HADAMARD4(a4, a5, a6, a7, had8y[4][i], had8y[5][i], had8y[6][i], had8y[7][i]);
                          b0  = abs2(a0 + a4) + abs2(a0 - a4);
                          b0 += abs2(a1 + a5) + abs2(a1 - a5);
                          b0 += abs2(a2 + a6) + abs2(a2 - a6);
                          b0 += abs2(a3 + a7) + abs2(a3 - a7);
                          sum += (sum_t)b0 + (b0 >> BITS_PER_SUM);  
                        } 
                      y_satd_shared[mode] = ( ((sum+2)>>2) << 8 | mode);
                  }    
                  if(ty==1){
                        sum2_t a0;
                        sum2_t a1;
                        sum2_t a2;
                        sum2_t a3;
                        sum2_t a4;
                        sum2_t a5;
                        sum2_t a6;
                        sum2_t a7;
                        sum2_t b0;

                        sum2_t sum =0;
                        for (int i = 0; i < 4; i++)
                        {
                          HADAMARD4(a0, a1, a2, a3, had8cr[0][i], had8cr[1][i], had8cr[2][i], had8cr[3][i]);
                          HADAMARD4(a4, a5, a6, a7, had8cr[4][i], had8cr[5][i], had8cr[6][i], had8cr[7][i]);
                          b0  = abs2(a0 + a4) + abs2(a0 - a4);
                          b0 += abs2(a1 + a5) + abs2(a1 - a5);
                          b0 += abs2(a2 + a6) + abs2(a2 - a6);
                          b0 += abs2(a3 + a7) + abs2(a3 - a7);
                          sum += (sum_t)b0 + (b0 >> BITS_PER_SUM);
                        }
                       cr_satd_shared[mode] = ( ((sum+2)>>2) << 8 | mode);
                  } 
                  if(ty==2){
                        sum2_t a0;
                        sum2_t a1;
                        sum2_t a2;
                        sum2_t a3;
                        sum2_t a4;
                        sum2_t a5;
                        sum2_t a6;
                        sum2_t a7;
                        sum2_t b0;

                        sum2_t sum = 0;
                        for (int i = 0; i < 4; i++)
                        {
                          HADAMARD4(a0, a1, a2, a3, had8cb[0][i], had8cb[1][i], had8cb[2][i], had8cb[3][i]);
                          HADAMARD4(a4, a5, a6, a7, had8cb[4][i], had8cb[5][i], had8cb[6][i], had8cb[7][i]);
                          b0  = abs2(a0 + a4) + abs2(a0 - a4);
                          b0 += abs2(a1 + a5) + abs2(a1 - a5);
                          b0 += abs2(a2 + a6) + abs2(a2 - a6);
                          b0 += abs2(a3 + a7) + abs2(a3 - a7);
                          sum += (sum_t)b0 + (b0 >> BITS_PER_SUM);
                        }
                      cb_satd_shared[mode] = ( ((sum+2)>>2) << 8 | mode);
                  }
             }
     } // end of bsize == 8

}// end of for
///////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////SATD 8 END/////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////    

    __syncthreads();

    if ( 0 == ty && 0 == tx )
    {

       int32_t *ptr_y_satd_shared = y_satd_shared;
       sort(ptr_y_satd_shared);
       extract(ptr_y_satd_shared, y_modes_shared);

       int offset = (blockIdx.y*gridDim.x + blockIdx.x)*35;

       for(int i=0;i<35;i++){
             res_y[offset] = ptr_y_satd_shared[i];
             y_modes[offset] = y_modes_shared[i];

             if(bx == 1 && by==0 && bsize==8){
                  printf("************GPU Result : satd value for index %d : mode %d has value %d\n",i,y_modes[offset], res_y[offset]);
             }
       }


/*
       int32_t *ptr_cr_satd_shared = cr_satd_shared;         
       sort(ptr_cr_satd_shared);
       extract(ptr_cr_satd_shared, res_cr, cr_modes);


       int32_t *ptr_cb_satd_shared = cb_satd_shared;
       sort(ptr_cb_satd_shared);
       extract(ptr_cb_satd_shared, res_cb, cb_modes);
*/

    }

    
} // End of kernel function hevcPredictionKernel()


